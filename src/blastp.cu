#include "hip/hip_runtime.h"
#include "blastp.h"
#include <assert.h>
#include <chrono>
#include <algorithm> 
#include <nvtx3/nvToolsExt.h>

#define PACK_KEY(k) ((k & ~0x7) | 0x3)

ThreadPool *pool;
mutex mu2;

// vector<SWResult> res_s[MAX_GROUPS_PER_ROUND][NUM_STREAM];

__constant__ uint32_t kHashTableCapacity_dev[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];
__constant__ uint32_t kHashTableOffset_dev[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];

__constant__ int SEED_LENGTH;
__constant__ int QIT_WIDTH;
__constant__ uint32_t MASK;

// 32 bit Murmur3 hash
inline __device__ uint32_t my_hash(uint32_t k, uint32_t kHashTableCapacity)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}

__device__ int insert_ot(KeyValue *hashtable, uint32_t kHashTableCapacity, uint32_t key, uint32_t value)
{
    key = PACK_KEY(key);
    uint32_t slot = my_hash(key, kHashTableCapacity);
    uint32_t b_slot = slot;
    while (true)
    {
        uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
        if (prev == kEmpty || prev == key)
        {
            // hashtable[slot].value = value;
            atomicAdd(&hashtable[slot].value, value);
            return 0;
        }
        slot = (slot + 1) & (kHashTableCapacity - 1);
        if (slot == b_slot)
        {
            return -1;
        }
    }
}

__global__ void seeding_kernel(KeyValue *ht, uint32_t *subj, size_t s_length_block, size_t s_length_total, const uint32_t *q_lengths, const int *q_num, const int *q_idx, int n_query, uint8_t *index_size_dev, uint32_t group_id)
{
    size_t s_begin = ((blockIdx.x * blockDim.x + threadIdx.x) * s_length_block) * 32;

    size_t s_len = s_length_block * 32;
    if (s_begin + s_len >= s_length_total - SEED_LENGTH)
        s_len = s_length_total - SEED_LENGTH - s_begin;
    if (s_len <= 0)
        return;

    size_t s_end = s_begin + s_len;

    for (size_t i = s_begin; i < s_end; i++)
    {
        size_t n_bit = i * 5;
        size_t pos = (n_bit >> 5);
        uint32_t mod = n_bit & 31;
        // assert(pos % 4 == 0);
        uint32_t qit_idx = (subj[pos] >> mod) & MASK;
        if (mod > (31 - (5 * SEED_LENGTH)))
        {
            qit_idx |= (subj[pos + 1] << (32 - mod)) & MASK;
        }

        int hit_size = index_size_dev[qit_idx];

        if (hit_size <= 0)
            continue;

        int qit_p = 0;
        for (int j = 0; j < hit_size; j++)
        {
            int pos = qit_idx * QIT_WIDTH + qit_p;
            int q_num_now = q_num[pos];
            int q_idx_now = q_idx[pos];
            if (q_num_now == -1)
            {
                qit_idx += q_idx_now;
                qit_p = 0;
                pos = qit_idx * QIT_WIDTH;
                q_num_now = q_num[pos];
                q_idx_now = q_idx[pos];
            }

            // printf("%d %d\n",q_num[qit_idx*qit_width+qit_p],q_idx[qit_idx*qit_width+qit_p]);
            unsigned int diag = q_lengths[q_num_now] + i - q_idx_now;
            // KeyValue *pHashTable_addr = ot + q_num_now * kHashTableCapacity_dev[q_num_now];
            KeyValue *pHashTable_addr = ht + kHashTableOffset_dev[group_id][q_num_now];
            int err = insert_ot(pHashTable_addr, kHashTableCapacity_dev[group_id][q_num_now], diag, 1);
            // assert(err != -1);
            if (err == -1)
            {
                printf("Voting Hash Table Full! G%uQ%uK%u\n", group_id, q_num_now, kHashTableCapacity_dev[group_id][q_num_now]);
            }
            qit_p++;
        }
    }
}

__global__ void filter_kernel(KeyValue *ht, Task *tasks, uint32_t *num_task, uint32_t *threshold, uint32_t group_id)
{
    uint32_t q_id = blockIdx.x;
    KeyValue *h_begin = ht + kHashTableOffset_dev[group_id][q_id];

    size_t each_length = (kHashTableCapacity_dev[group_id][q_id] - 1) / blockDim.x + 1;
    h_begin += each_length * threadIdx.x;
    KeyValue *h_end = h_begin + each_length;

    KeyValue *total_end = ht + kHashTableOffset_dev[group_id][q_id] + kHashTableCapacity_dev[group_id][q_id];
    h_end = h_end > total_end ? total_end : h_end;

    Task *task_begin = tasks;

    for (KeyValue *kv = h_begin; kv < h_end; kv++)
    {
        if (kv->key != kEmpty && kv->value != kEmpty && kv->value >= threshold[q_id])
        {
            uint32_t idx = atomicAdd(num_task, 1);
            if (idx >= MAX_FILTER_TASK)
            {
                printf("Filter Task Vector Full! G%uQ%uT%u\n", group_id, q_id, idx);
                return;
            }
            task_begin[idx].key = kv->key;
            task_begin[idx].value = kv->value;
            task_begin[idx].q_id = q_id;
        }
    }

    // size_t total_length = kHashTableOffset_dev[group_id][n_query-1] + kHashTableCapacity_dev[group_id][n_query-1];
    // size_t each_length = (total_length-1)/b + 1;
}


__global__ void multibatch_banded_sw_kernel(uint32_t* totalTasks,uint32_t* q_lens, uint32_t* q_idxs, Task* task,
                const char* q, const char* c, size_t c_len,
                int * score_d,
                // int* q_len_d,int* s_len_d,
                size_t* q_end_d, size_t* s_end_d,
                char* cigar_op_d, int* cigar_cnt_d,int* cigar_len_d,
                int *rd, record* rt_d,int band_width,
                const int* BLOSUM62_d)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    const int* BLOSUM62 = BLOSUM62_d;
    record* rt = rt_d + idx * MaxBW * (TILE_SIZE + 1);
    
    for (size_t global_idx = idx; global_idx < (*totalTasks); global_idx += BatchSize) {
        
        size_t n = q_lens[task[global_idx].q_id];
        if(n > MaxQueryLen) printf("## Query Len %ld\n", n);
        assert( n < MaxQueryLen);
    
        size_t q_idx  = q_idxs[task[global_idx].q_id];
        size_t diag  = task[global_idx].key;
        
        int64_t c_begin = (int64_t)diag - band_width - n + 2;
        size_t c_end = diag + band_width;
       
        // size_t width = 2 * band_width + 1;
        size_t height = MaxQueryLen + 1;
        // assert(width < MaxBW);
        
        size_t t_height = TILE_SIZE + 1;
        
        // record *rt = (record *)malloc(width * t_height * sizeof(record));
        // memset(rt, 0, width * t_height * sizeof(record));

        size_t max_q = 0;
        size_t max_c = 0;
        int score = 0, Score = 0;
        // cal maxScore and it's position
        for (size_t it = 0; it * TILE_SIZE < n; it++) {
            
            size_t q_offset = it * TILE_SIZE;

            for(size_t _q = 0; _q < t_height-1 && q_offset + _q < n; ++_q){
                for(size_t _c = 0; _c < 2 * band_width - 1; ++_c){
                    
                    if(c_begin + _c+ q_offset + _q < 0) continue;
                    if(c_begin + _c+ q_offset + _q >= c_len) break;

                    char chq = q[q_idx + q_offset + _q];
                    char chc = get_char_d(c, c_begin + q_offset + _c + _q);
                    
                    if (chq == END_SIGNAL || chc == END_SIGNAL)
                    {
                        continue;
                    }
                    //rt(_q,_c) -> (_q+1) * width + _c + 1
                    // logical m(_q,_c).x = max(m(_q-1,_c).x + SCORE_GAP_EXT, m(_q-1,_c).m +SCORE_GAP, 0 );
                    // logical m(_q,_c).y = max(m(_q,_c-1).y + SCORE_GAP_EXT, m(_q,_c-1).m +SCORE_GAP, 0 );
                    // logical m(_q,_c).m = max(m(_q-1,_c-1).y,m(_q-1,_c-1).x,m(_q-1,_c-1).m, 0 );
                    
                    rt[calIndex(_q,_c,MaxBW)].x = max3(rt[calTop(_q,_c,MaxBW)].x + SCORE_GAP_EXT,  rt[calTop(_q,_c,MaxBW)].m + SCORE_GAP, 0);
                    rt[calIndex(_q,_c,MaxBW)].y = max3(rt[calLeft(_q,_c,MaxBW)].y + SCORE_GAP_EXT, rt[calLeft(_q,_c,MaxBW)].m + SCORE_GAP, 0);

                    if (chq == ILLEGAL_WORD || chc == ILLEGAL_WORD)
                    {
                        // illegal word
                        rt[calIndex(_q,_c,MaxBW)].m = 0;
                    }
                    else
                    {
                        rt[calIndex(_q,_c,MaxBW)].m = max2(max3(rt[calDiag(_q,_c,MaxBW)].x, rt[calDiag(_q,_c,MaxBW)].y, rt[calDiag(_q,_c,MaxBW)].m) + BLOSUM62[chq * 26 + chc], 0);
                    }

                    score = max3(rt[calIndex(_q,_c,MaxBW)].x, rt[calIndex(_q,_c,MaxBW)].y, rt[calIndex(_q,_c,MaxBW)].m);
                    
                    // printf("(q = %c,c = %c) BLOSUM62 = %d rt[_q * width + _c].s = %d\n", chq+65,chc+65,BLOSUM62[chq * 26 + chc], rt[_q * width + _c].s);
                    // (rd + global_idx*direct_matrixSize)[_c * height + _q + q_offset] = (score == rt[_q * width + _c].x)*TOP + (score == rt[_q * width + _c].y)*LEFT + (rt[_c * height + _q + q_offset].m)*DIAG; 
                    
                    rd[calIndex(_c, _q+q_offset,height) * BatchSize + idx] = (score?( \
                        (score == rt[calIndex(_q,_c,MaxBW)].m) ? DIAG : \
                        ((score == rt[calIndex(_q,_c,MaxBW)].y) ? LEFT :TOP )):0);
                    
                    if (Score < score)
                    {
                        Score = score;
                        max_c = _c;
                        max_q = _q + q_offset;
                    }
                    // printf("(q = %c,c = %c) score = %d maxScore = %d direction = %d\n", chq+65,chc+65,r[_q*width + _c].s,r[max_c * height + max_q].s,r[_q * width + _c].d);
                }
            }
            memcpy(rt,rt + (t_height - 1) * MaxBW ,MaxBW * sizeof(record));
            // Hit when target is not long enough, there are some cells should be zero
            memset(rt + MaxBW, 0, (t_height - 1) * MaxBW * sizeof(record));

        }

        score_d[global_idx] = Score;
        // res[global_idx].score = Score;
        if(Score == 0){
            printf("## it = %d\n",global_idx);
        }
        assert(Score != 0);

        size_t cur_q= max_q;
        size_t cur_c = max_c;

        q_end_d[global_idx] = cur_q + q_idx;
        s_end_d[global_idx] = c_begin + cur_c + cur_q;

        int cnt_q = 0, cnt_c = 0;
        int cigar_len = 0;
        assert(rd[BatchSize * calIndex(cur_c,cur_q,height) + idx] != 0);
        while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx])
        {
            int d = rd[BatchSize * calIndex(cur_c,cur_q,height) + idx];
            // size_t res_q = (d&0x01) ? (cur_q + q_idx) : (size_t)-1;
            // size_t res_c = (d&0x02) ? (c_begin + cur_c + cur_q) : (size_t)-1;
            
            // q_res_d[global_idx* MaxAlignLen + (cnt_q)] = (res_q);
            // s_res_d[global_idx* MaxAlignLen + (cnt_c)] = (res_c);
            int cur_cigar_cnt = 0;
            while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx] && rd[BatchSize * calIndex(cur_c,cur_q,height) + idx]==d){
                cur_cigar_cnt ++;
                
                //TOP 01b, left 10b, diag 11b
                //DIAG : cur_q -= 1
                //TOP : cur_q -= 1, cur_c += 1;
                //LEFT : cur_c -= 1
                cur_q -= (d == DIAG || d == TOP);
                cur_c += (d == TOP); // Increment cur_c if TOP (01b)
                cur_c -= (d == LEFT); // Decrement cur_c if LEFT (10b)
            }
            (cigar_cnt_d + global_idx * MaxAlignLen)[cigar_len] = cur_cigar_cnt;
            (cigar_op_d + global_idx * MaxAlignLen)[cigar_len++] = ((d==DIAG)?'M':((d==TOP)?'D':'I'));
        }

        // free(rt);
        assert(cigar_len > 0);
        cigar_len_d[global_idx] = cigar_len;
        
        memset(rt,0, MaxBW * (TILE_SIZE + 1) * sizeof(record));
        for(size_t i = 0; i < MaxBW; ++ i){
            for(size_t j = 0; j < MaxQueryLen+1; ++ j){
                rd[BatchSize * (i * (MaxQueryLen+1) + j) + idx] = 0;
            }
        }

    }
}
__global__ void banded_sw_kernel(
                int NumTasks,
                uint32_t* q_lens, uint32_t* q_idxs, Task* task,
                const char* q, const char* c, size_t c_len,
                int * score_d,
                size_t* q_end_d, size_t* s_end_d,
                char* cigar_op_d, int* cigar_cnt_d,int* cigar_len_d,
                int *rd, record* rt_d,int band_width,
                int* BLOSUM62_d){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx >= NumTasks) return;

    size_t n = q_lens[task[idx].q_id];
    if(n > MaxQueryLen) printf("## Query Len %ld\n", n);
    assert( n < MaxQueryLen);
   
    size_t q_idx  = q_idxs[task[idx].q_id];
    size_t diag  = task[idx].key;
    
    int64_t c_begin = (int64_t)diag - band_width - n + 2;
    // size_t c_end = diag + band_width;
    int* BLOSUM62 = BLOSUM62_d;
    record* rt = rt_d + idx * MaxBW * (TILE_SIZE + 1);
    
    // __shared__ int shared_BLOSUM62[26 * 26];
    
    // for(size_t i = 0; i < MaxBW; ++ i){
    //     for(size_t j = 0; j < MaxQueryLen+1; ++ j){
    //         rd[BatchSize * (i * (MaxQueryLen+1) + j) + idx] = 0;
    //     }
    // }
    // __syncthreads(); // Wait for the copy to compl .ete


    size_t width = 2 * band_width + 1;
    size_t height = MaxQueryLen + 1;
    assert(width < MaxBW);

    //init:
    score_d[idx] = 0;
    
    size_t t_height = TILE_SIZE + 1;
    
    // record *rt = (record *)malloc(width * t_height * sizeof(record));
    // memset(rt, 0, width * t_height * sizeof(record));

    size_t max_q = 0;
    size_t max_c = 0;
    int score = 0, Score = 0;
    // cal maxScore and it's position
    for (size_t it = 0; it * TILE_SIZE < n; it++) {
        
        size_t q_offset = it * TILE_SIZE;

        for(size_t _q = 0; _q < t_height-1 && q_offset + _q < n; ++_q){
            for(size_t _c = 0; _c < width-2; ++_c){
                
                if(c_begin + _c+ q_offset + _q < 0) continue;
                if(c_begin + _c+ q_offset + _q >= c_len) break;

                char chq = q[q_idx + q_offset + _q];
                char chc = get_char_d(c, c_begin + q_offset + _c + _q);
                
                if (chq == END_SIGNAL || chc == END_SIGNAL)
                {
                    continue;
                }
                //rt(_q,_c) -> (_q+1) * width + _c + 1
                // logical m(_q,_c).x = max(m(_q-1,_c).x + SCORE_GAP_EXT, m(_q-1,_c).m +SCORE_GAP, 0 );
                // logical m(_q,_c).y = max(m(_q,_c-1).y + SCORE_GAP_EXT, m(_q,_c-1).m +SCORE_GAP, 0 );
                // logical m(_q,_c).m = max(m(_q-1,_c-1).y,m(_q-1,_c-1).x,m(_q-1,_c-1).m, 0 );
                
                rt[calIndex(_q,_c,MaxBW)].x = max3(rt[calTop(_q,_c,MaxBW)].x + SCORE_GAP_EXT,  rt[calTop(_q,_c,MaxBW)].m + SCORE_GAP, 0);
                rt[calIndex(_q,_c,MaxBW)].y = max3(rt[calLeft(_q,_c,MaxBW)].y + SCORE_GAP_EXT, rt[calLeft(_q,_c,MaxBW)].m + SCORE_GAP, 0);

                if (chq == ILLEGAL_WORD || chc == ILLEGAL_WORD)
                {
                    // illegal word
                    rt[calIndex(_q,_c,MaxBW)].m = 0;
                }
                else
                {
                    rt[calIndex(_q,_c,MaxBW)].m = max2(max3(rt[calDiag(_q,_c,MaxBW)].x, rt[calDiag(_q,_c,MaxBW)].y, rt[calDiag(_q,_c,MaxBW)].m) + BLOSUM62[chq * 26 + chc], 0);
                }

                score = max3(rt[calIndex(_q,_c,MaxBW)].x, rt[calIndex(_q,_c,MaxBW)].y, rt[calIndex(_q,_c,MaxBW)].m);
                
                // printf("(q = %c,c = %c) BLOSUM62 = %d rt[_q * width + _c].s = %d\n", chq+65,chc+65,BLOSUM62[chq * 26 + chc], rt[_q * width + _c].s);
                // (rd + idx*direct_matrixSize)[_c * height + _q + q_offset] = (score == rt[_q * width + _c].x)*TOP + (score == rt[_q * width + _c].y)*LEFT + (rt[_c * height + _q + q_offset].m)*DIAG; 
            
                rd[calIndex(_c, _q+q_offset,height) * BatchSize + idx] = (score?( \
                    (score == rt[calIndex(_q,_c,MaxBW)].m) ? DIAG : \
                    ((score == rt[calIndex(_q,_c,MaxBW)].y) ? LEFT :TOP )):0);
                
                if (Score < score)
                {
                    Score = score;
                    max_c = _c;
                    max_q = _q + q_offset;
                }
                // printf("(q = %c,c = %c) score = %d maxScore = %d direction = %d\n", chq+65,chc+65,r[_q*width + _c].s,r[max_c * height + max_q].s,r[_q * width + _c].d);
            }
        }
        memcpy(rt,rt + (t_height - 1) * MaxBW ,MaxBW * sizeof(record));
        // Hit when target is not long enough, there are some cells should be zero
        memset(rt + MaxBW, 0, (t_height - 1) * MaxBW * sizeof(record));

    }

    score_d[idx] = Score;
    // res[idx].score = Score;
    assert(Score != 0);

    size_t cur_q= max_q;
    size_t cur_c = max_c;

    q_end_d[idx] = cur_q + q_idx;
    s_end_d[idx] = c_begin + cur_c + cur_q;

    // int cnt_q = 0, cnt_c = 0;
    int cigar_len = 0;
    while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx])
    {
        int d = rd[BatchSize * calIndex(cur_c,cur_q,height) + idx];
        // size_t res_q = (d&0x01) ? (cur_q + q_idx) : (size_t)-1;
        // size_t res_c = (d&0x02) ? (c_begin + cur_c + cur_q) : (size_t)-1;
        
        // q_res_d[idx* MaxAlignLen + (cnt_q)] = (res_q);
        // s_res_d[idx* MaxAlignLen + (cnt_c)] = (res_c);
        int cur_cigar_cnt = 0;
        while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx] && rd[BatchSize * calIndex(cur_c,cur_q,height) + idx]==d){
            cur_cigar_cnt ++;
            
            //TOP 01b, left 10b, diag 11b
            //DIAG : cur_q -= 1
            //TOP : cur_q -= 1, cur_c += 1;
            //LEFT : cur_c -= 1
            cur_q -= (d == DIAG || d == TOP);
            cur_c += (d == TOP); // Increment cur_c if TOP (01b)
            cur_c -= (d == LEFT); // Decrement cur_c if LEFT (10b)
        }
        (cigar_cnt_d + idx * MaxAlignLen)[cigar_len] = cur_cigar_cnt;
        (cigar_op_d + idx * MaxAlignLen)[cigar_len++] = ((d==DIAG)?'M':((d==TOP)?'D':'I'));
    }

    // free(rt);
    // printf("@@ cigar_len %d %d\n", idx,cigar_len);
    assert(cigar_len > 0);
    cigar_len_d[idx] = cigar_len;
}

#ifdef USE_GPU_SW
void handle_results(hipEvent_t &stream, Task *task_host, uint32_t *num_task, QueryGroup &q_group, size_t s_length, int stream_id, vector<SWResult> &res, SWTasks &sw_task)
{
    hipEventSynchronize(stream);
    mu2.lock();
    cout << "=";
    res.clear();
    size_t n_task_pre = sw_task.num_task;
    size_t n_task = sw_task.num_task + *num_task;
    sw_task.c_len += s_length;
    sw_task.q_idxs.resize(n_task);
    sw_task.q_lens.resize(n_task);
    sw_task.q_len4_offs.resize(n_task+1);
    sw_task.s_len4_offs.resize(n_task+1);
    sw_task.diags.resize(n_task);
    sw_task.info.resize(n_task);
    Task *t_begin = task_host;
    sw_task.num_task = n_task;
    res.resize(*num_task);
#pragma omp parallel for
    for (int i = 0; i < *num_task; i++)
    {
        Task &kv = *(t_begin + i);
        sw_task.q_idxs[i + n_task_pre]=q_group.offset[kv.q_id];
        sw_task.q_lens[i + n_task_pre]=q_group.length[kv.q_id];
        sw_task.diags[i + n_task_pre] =  sw_task.c_offset + kv.key;
        sw_task.info[i+ n_task_pre].group_id = q_group.group_id;
        sw_task.info[i+ n_task_pre].stream_id = stream_id;
        sw_task.info[i+ n_task_pre].idx = i;
        res[i].num_q = kv.q_id;
    }

    for (int i = 0; i < *num_task; i++)
    {
        int q_len4 = sw_task.q_lens[i+ n_task_pre];
        q_len4 = q_len4 % 4? q_len4 + (4 - (q_len4 % 4)) : q_len4;
        int s_len4 = sw_task.q_lens[i+ n_task_pre] + (band_width << 1);
        s_len4 = s_len4 %4? s_len4 + (4-(s_len4%4)):s_len4;
        sw_task.q_len4_offs[i+ n_task_pre+1] = sw_task.q_len4_offs[i+ n_task_pre] + q_len4;
        sw_task.s_len4_offs[i+ n_task_pre+1] = sw_task.s_len4_offs[i+ n_task_pre] + s_len4;
    }

    sw_task.c_offset += s_length;
    mu2.unlock();

}
#else
void handle_results(hipEvent_t &stream, const char *query, const char *subj, Task *task_host, uint32_t *num_task, QueryGroup &q_group, size_t s_length, int stream_id, vector<SWResult> &res, SWTasks &sw_task, ThreadPool *pool, vector<future<int>> &rs)
{
    hipEventSynchronize(stream);
    cout << "=";
    res.clear();
    res.resize(*num_task);
    sw_task.q = query;
    sw_task.c = subj;
    sw_task.c_len = s_length;
    sw_task.q_idxs.resize(*num_task);
    sw_task.q_lens.resize(*num_task);
    sw_task.diags.resize(*num_task);
    Task *t_begin = task_host;
    sw_task.num_task = *num_task;
#pragma omp parallel for
    for (int i = 0; i < *num_task; i++)
    {
        Task &kv = *(t_begin + i);
        sw_task.q_idxs[i]=q_group.offset[kv.q_id];
        sw_task.q_lens[i]=q_group.length[kv.q_id];
        sw_task.diags[i]=kv.key;
        res[i].num_q = kv.q_id;
    }
    mu2.lock();
    for (int i = 0; i < sw_task.num_task; ++i)
    {
        rs.emplace_back(pool->enqueue([&, i]
                                      {
            smith_waterman_kernel(i,&res[i],&sw_task);
            return i; }));
    }
    mu2.unlock();
}
#endif
struct StreamState {
    std::atomic<bool> callbackCompleted;
    StreamState() : callbackCompleted(false) {}
};


typedef struct CallbackData{
    StreamState* streamState;
    uint32_t n;
    int s;
    size_t* g_begin;
    size_t off_g;
    int* cigar_len_h;
    char* cigar_op_h;
    int* cigar_cnt_h;
    size_t* q_end_h;
    size_t* s_end_h;
    vector<SWResult>& res_s;
    Task* task_host;
    const char* query;
    const char* subj;
    int* score_h;
}CallbackData;

void call_results(hipEvent_t& event, \
                hipStream_t& stream,
                size_t* q_end_d, size_t* s_end_d,
                char* cigar_op_d, int* cigar_cnt_d,int* cigar_len_d,
                const char* query, const char* subj,
                Task* task_host, int num_task,
                int* cigar_len_h, char* cigar_op_h, int* cigar_cnt_h,
                size_t* q_end_h, size_t* s_end_h,
                int* score_h,
                std::vector<SWResult>& res_s, int begin,
                ThreadPool* pool, std::vector<std::future<int>>& rs) {
    CUDA_CALL(hipEventSynchronize(event));
    CUDA_CALL(hipEventDestroy(event));
    CUDA_CALL(hipFreeAsync(s_end_d, stream));
    CUDA_CALL(hipFreeAsync(q_end_d, stream));
    CUDA_CALL(hipFreeAsync(cigar_op_d, stream));
    CUDA_CALL(hipFreeAsync(cigar_cnt_d, stream));
    CUDA_CALL(hipFreeAsync(cigar_len_d, stream));
    // cout << "=";

    for (size_t i = 0; i < num_task; ++i) {
        // rs.emplace_back(pool->enqueue([&, i] {
            
            cigar_to_index_and_report(i, begin, cigar_len_h, cigar_op_h, cigar_cnt_h,
                        q_end_h, s_end_h, res_s, score_h, task_host, query, subj);
            
        //     return static_cast<int>(i);
        // }));
    }
}
// void CUDART_CB hostCallBack(hipStream_t stream, hipError_t status,void* userData) {
//     CallbackData* data = static_cast<CallbackData*>(userData);
//     data->res_s.resize(data->n);
//     *(data->g_begin) =  *(data->g_begin) + data->off_g;
// #pragma omp parallel for
//     for (size_t i = 0; i < data->n; ++i) {
//         assert(data->cigar_len_h[i] < MaxAlignLen && data->cigar_len_h[i]);
//         //TODO From cigar to index
//         cigar_to_index(data->cigar_len_h[i],
//                        data->cigar_op_h + i * MaxAlignLen,
//                        data->cigar_cnt_h + i * MaxAlignLen,
//                        data->q_end_h[i], data->s_end_h[i],
//                        ref(data->res_s[i].q_res),
//                        ref(data->res_s[i].s_res));
       
//         data->res_s[i].score = data->score_h[i];            
//         data->res_s[i].num_q = data->task_host[i].q_id;
//         generate_report(&data->res_s[i], data->query, data->subj);
//     }
//     data->streamState->callbackCompleted.store(true, std::memory_order_release);
// }


void search_db_batch(const char *query, char *subj[], vector<QueryGroup> &q_groups, size_t s_length[], Task *task_host[][NUM_STREAM], uint32_t *task_num_host[][NUM_STREAM], size_t max_hashtable_capacity, uint32_t max_n_query, uint32_t total_len_query, string db_name, uint32_t db_num, vector<SWResult> *res, size_t total_db_size, TimeProfile &time_prof)
{
    struct timeval t_start, t_end, tt_start;

    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(SEED_LENGTH), &seed_length, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(QIT_WIDTH), &qit_width, sizeof(int)));
    uint32_t mask = (uint32_t)pow(2, 5 * seed_length) - 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MASK), &mask, sizeof(uint32_t)));

    size_t sum_s_len = 0;
    for (int i = 0; i < NUM_STREAM; i++)
    {
        sum_s_len += s_length[i];
        assert(s_length[i] % 32 == 0);
    }

    char *subj_dev;
    CUDA_CALL(hipMalloc((void **)&subj_dev, sum_s_len / 8 * 5));

    int n_groups = q_groups.size();
    if (n_groups > MAX_GROUPS_PER_ROUND)
        n_groups = MAX_GROUPS_PER_ROUND;

    int *q_num_dev[n_groups];
    int *q_idx_dev[n_groups];
    uint32_t *q_lengths_dev[n_groups];
    uint8_t *index_size_dev[n_groups];
    uint32_t *threshold_dev[n_groups];
    uint32_t *q_offset_dev[n_groups];

    uint32_t kHashTableCapacity_host[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];
    uint32_t kHashTableOffset_host[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];

    for (int g = 0; g < n_groups; g++)
    {
        CUDA_CALL(hipMalloc((void **)&q_num_dev[g], qit_length * qit_width * sizeof(int)));
        CUDA_CALL(hipMalloc((void **)&q_idx_dev[g], qit_length * qit_width * sizeof(int)));
        CUDA_CALL(hipMalloc((void **)&q_lengths_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
        CUDA_CALL(hipMalloc((void **)&q_offset_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
        CUDA_CALL(hipMalloc((void **)&index_size_dev[g], qit_length * sizeof(uint8_t)));
        CUDA_CALL(hipMalloc((void **)&threshold_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
    }
    KeyValue *pHashTable[n_groups][NUM_STREAM];
    Task *task_dev[n_groups][NUM_STREAM];
    uint32_t *task_num_dev[n_groups][NUM_STREAM];

    for (int g = 0; g < n_groups; g++)
    {
        for (int s = 0; s < NUM_STREAM; s++)
        {
            // pHashTable[s] = create_hashtable(max_hashtable_capacity);
            CUDA_CALL(hipMalloc((void **)&task_dev[g][s], MAX_FILTER_TASK * sizeof(Task)));
            CUDA_CALL(hipMemset(task_dev[g][s], 0, MAX_FILTER_TASK * sizeof(Task)));
            CUDA_CALL(hipMalloc((void **)&task_num_dev[g][s], sizeof(uint32_t)));
            CUDA_CALL(hipMemset(task_num_dev[g][s], 0, sizeof(uint32_t)));
        }
    }

    char *s_name[NUM_STREAM] = {0};
    size_t *s_offsets[NUM_STREAM] = {0};
    size_t *sn_offsets[NUM_STREAM] = {0};
    size_t s_num[NUM_STREAM] = {0};

    int mingridsize_seeding, mingridsize_filter;
    int threadblocksize_seeding, threadblocksize_filter;
    CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&mingridsize_seeding, &threadblocksize_seeding, seeding_kernel, 0, 0));
    CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&mingridsize_filter, &threadblocksize_filter, filter_kernel, 0, 0));

    // cout << "Seeding Block size:" << threadblocksize_seeding <<"," << mingridsize_seeding <<endl;
    // cout << "Filter Block size:" << threadblocksize_filter <<"," << mingridsize_filter <<endl;

    size_t free_byte, total_byte;
    CUDA_CALL(hipMemGetInfo(&free_byte, &total_byte));
    cout << "GPU mem: " << (double)(total_byte - free_byte) / (1073741824) << " GB / " << (double)total_byte / (1073741824) << " GB" << endl;

#ifndef USE_GPU_SW
    SWTasks sw_tasks[q_groups.size()][NUM_STREAM];
#endif
    SWTasks sw_tasks_total;
    vector<SWResult> res_s[q_groups.size()][NUM_STREAM];

    char* query_dev;
    CUDA_CALL(hipMalloc((void **)&query_dev, total_len_query));
    CUDA_CALL(hipMemcpy(query_dev, query, total_len_query, hipMemcpyHostToDevice));
#ifdef USE_GPU_SW
    sw_tasks_total.q = query;
#endif

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);

#ifdef USE_GPU_DIFFUSE
    int direct_matrixSize = (MaxQueryLen+1) * MaxBW;
    int threadsPerBlock = 64;  // 根据 shared memory 限制调整
    int blocks = (BatchSize + threadsPerBlock - 1) / threadsPerBlock;
    
    int* rd[n_groups][NUM_STREAM][MaxNumBatch];   // direct_matrixSize * BatchSize * sizeof(int)
    record* rt[n_groups][NUM_STREAM][MaxNumBatch];
    int* BLOSUM62_d;
    int* score_d[n_groups][NUM_STREAM][MaxNumBatch], *score_h[n_groups][NUM_STREAM][MaxNumBatch];
    
    size_t* q_end_d[n_groups][NUM_STREAM][MaxNumBatch], *q_end_h[n_groups][NUM_STREAM][MaxNumBatch];
    size_t* s_end_d[n_groups][NUM_STREAM][MaxNumBatch], *s_end_h[n_groups][NUM_STREAM][MaxNumBatch];
    char* cigar_op_d[n_groups][NUM_STREAM][MaxNumBatch], *cigar_op_h[n_groups][NUM_STREAM][MaxNumBatch];
    int *cigar_cnt_d[n_groups][NUM_STREAM][MaxNumBatch], *cigar_cnt_h[n_groups][NUM_STREAM][MaxNumBatch];
    int* cigar_len_d[n_groups][NUM_STREAM][MaxNumBatch], *cigar_len_h[n_groups][NUM_STREAM][MaxNumBatch];


    for(int g = 0; g < n_groups; ++ g){

        for (int s = 0; s < NUM_STREAM; s++)
        {
            for(int cur = 0; cur < MaxNumBatch; ++ cur){

                CUDA_CALL(hipHostMalloc(&score_h[g][s][cur], sizeof(int) * BatchSize));
            
                CUDA_CALL(hipHostMalloc(&q_end_h[g][s][cur], sizeof(size_t) * BatchSize));
                CUDA_CALL(hipHostMalloc(&s_end_h[g][s][cur], sizeof(size_t) * BatchSize));
                
                CUDA_CALL(hipHostMalloc(&cigar_cnt_h[g][s][cur], sizeof(int) * MaxAlignLen * BatchSize));
                CUDA_CALL(hipHostMalloc(&cigar_op_h[g][s][cur], sizeof(char) * MaxAlignLen * BatchSize));
                CUDA_CALL(hipHostMalloc(&cigar_len_h[g][s][cur], sizeof(int)* BatchSize));
        
            }
        }
    }
#endif

    int g_begin = 0;

    size_t s_begin = 0;
    hipStream_t streams;
    hipStream_t malloc_streams;
    CUDA_CALL(hipStreamCreate(&streams));
    CUDA_CALL(hipStreamCreate(&malloc_streams));
    CUDA_CALL(hipMallocAsync((void**)&BLOSUM62_d, 26 * 26 * sizeof(int), malloc_streams));
    CUDA_CALL(hipMemcpyAsync(BLOSUM62_d, BLOSUM62, 26 * 26 * sizeof(int),hipMemcpyHostToDevice,malloc_streams));
    vector<size_t> s_begin_vec;
#ifdef USE_GPU_DIFFUSE
    hipStream_t copy_streams[MaxNumBatch];
    for(int cur = 0; cur < MaxNumBatch; ++ cur){
        hipStreamCreate(&copy_streams[cur]);
    }
#endif
    for(int s = 0; s < NUM_STREAM; ++ s){
        s_begin_vec.push_back(s_begin);
        size_t s_length_stream = s_length[s];
        size_t s_length_stream_byte = s_length_stream / 8 * 5;
        s_begin += s_length_stream_byte;
        // printf("start stream %d\n", s);
        size_t s_length_stream_block = s_length_stream / 32 * 5;
        size_t each_length_block = (s_length_stream_block - 1) / (mingridsize_seeding * threadblocksize_seeding) + 1;
        CUDA_CALL(hipMemcpyAsync(subj_dev + s_begin_vec[s], subj[s], s_length_stream_byte, hipMemcpyHostToDevice, malloc_streams));

    }
    for (int g_begin = 0; g_begin < q_groups.size(); g_begin += MAX_GROUPS_PER_ROUND)
    {
        sw_tasks_total.c_offset = 0;
        double group_time = 0;
        cout << "Group " << g_begin + 1 << "/" << q_groups.size() << "\t[";
        gettimeofday(&t_start, NULL);
        n_groups = q_groups.size() - g_begin;
        if (n_groups > MAX_GROUPS_PER_ROUND)
            n_groups = MAX_GROUPS_PER_ROUND;
        for (int g = g_begin; g < g_begin + n_groups; g++)
        {
            int g_idx = g - g_begin;
            CUDA_CALL(hipMemcpy(q_num_dev[g_idx], q_groups[g].qit.q_num, qit_length * qit_width * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_idx_dev[g_idx], q_groups[g].qit.q_idx, qit_length * qit_width * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(index_size_dev[g_idx], q_groups[g].qit.index_size, qit_length * sizeof(uint8_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_lengths_dev[g_idx], q_groups[g].length, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_offset_dev[g_idx], q_groups[g].offset, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(threshold_dev[g_idx], q_groups[g].min_diag_hit, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            memcpy(kHashTableCapacity_host[g_idx], q_groups[g].hashtable_capacity, sizeof(uint32_t) * MAX_QUERY_PER_GROUP);
            memcpy(kHashTableOffset_host[g_idx], q_groups[g].hashtable_offset, sizeof(uint32_t) * MAX_QUERY_PER_GROUP);
        }

        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(kHashTableCapacity_dev), kHashTableCapacity_host, sizeof(uint32_t) * MAX_QUERY_PER_GROUP * MAX_GROUPS_PER_ROUND));
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(kHashTableOffset_dev), kHashTableOffset_host, sizeof(uint32_t) * MAX_QUERY_PER_GROUP * MAX_GROUPS_PER_ROUND));

        
        thread result_threads[n_groups][NUM_STREAM];
#ifdef USE_GPU_DIFFUSE
        
        hipEvent_t copies_done[n_groups][NUM_STREAM][MaxNumBatch];
        hipEvent_t kernels_done[n_groups][NUM_STREAM][MaxNumBatch];
        
        thread report_threads[n_groups][NUM_STREAM][MaxNumBatch];
        vector<future<int>> rs_report[n_groups][NUM_STREAM][MaxNumBatch];

#endif
        hipEvent_t malloc_finished[n_groups][NUM_STREAM];
        hipEvent_t seeding_finished[n_groups][NUM_STREAM];
        
#ifndef USE_GPU_SW
        vector<future<int>> rs[n_groups][NUM_STREAM];
#endif
        gettimeofday(&t_end, NULL);
        group_time += timeuse(t_start, t_end);
        time_prof.mem_time += timeuse(t_start, t_end);
        // cout << "Prepare mem and data Time: " << timeuse(t_start, t_end) << endl;

        gettimeofday(&t_start, NULL);
        nvtxRangePush("MyProfileRegion");
        for (int g = g_begin; g < g_begin + n_groups; g++)
        {
            int g_idx = g - g_begin;

            for (int s = 0; s < NUM_STREAM; s++)
            {
                CUDA_CALL(hipEventCreate(&malloc_finished[g_idx][s]));
                CUDA_CALL(hipEventCreate(&seeding_finished[g_idx][s]));
#ifdef USE_GPU_DIFFUSE
                for(int cur = 0; cur < MaxNumBatch; ++ cur){
                    CUDA_CALL(hipEventCreate(&copies_done[g_idx][s][cur]));
                    CUDA_CALL(hipEventCreate(&kernels_done[g_idx][s][cur]));
                }
#endif
                // pHashTable[g_idx][s] = create_hashtable(max_hashtable_capacity);
                CUDA_CALL(hipStreamSynchronize(malloc_streams));
                pHashTable[g_idx][s] = create_hashtable_async(max_hashtable_capacity,malloc_streams);
                size_t s_length_stream = s_length[s];
                size_t s_length_stream_byte = s_length_stream / 8 * 5;
                size_t s_length_stream_block = s_length_stream / 32 * 5;
                size_t each_length_block = (s_length_stream_block - 1) / (mingridsize_seeding * threadblocksize_seeding) + 1;

                int n_query = q_groups[g].n_query;
                if (g > 0)
                {
                    CUDA_CALL(hipMemsetAsync(task_dev[g_idx][s], 0, MAX_FILTER_TASK * sizeof(Task), malloc_streams));
                    CUDA_CALL(hipMemsetAsync(task_num_dev[g_idx][s], 0, sizeof(uint32_t), malloc_streams));
                    CUDA_CALL(hipMemsetAsync(pHashTable[g_idx][s], 0xff, max_hashtable_capacity * sizeof(KeyValue), malloc_streams));
                }
                // assert(s_begin_vec[s] == s_begin);
                CUDA_CALL(hipEventRecord(malloc_finished[g_idx][s], malloc_streams));
                CUDA_CALL(hipEventSynchronize((malloc_finished[g_idx][s])));
                CUDA_CALL(hipEventDestroy(malloc_finished[g_idx][s]));

                seeding_kernel<<<mingridsize_seeding, threadblocksize_seeding, 0, streams>>>(pHashTable[g_idx][s], (uint32_t *)(subj_dev + s_begin_vec[s]), each_length_block, s_length_stream, q_lengths_dev[g_idx], q_num_dev[g_idx], q_idx_dev[g_idx], n_query, index_size_dev[g_idx], g_idx);
                filter_kernel<<<n_query, threadblocksize_filter, 0, streams>>>(pHashTable[g_idx][s], task_dev[g_idx][s], task_num_dev[g_idx][s], threshold_dev[g_idx], g_idx);
                
                // destroy_hashtable(pHashTable[g_idx][s]);
                destroy_hashtable_async(pHashTable[g_idx][s],malloc_streams);
                
                CUDA_CALL(hipMemcpyAsync(task_num_host[g_idx][s], task_num_dev[g_idx][s], sizeof(uint32_t), hipMemcpyDeviceToHost, streams));    
                CUDA_CALL(hipMemcpyAsync(task_host[g_idx][s], task_dev[g_idx][s], MAX_FILTER_TASK * sizeof(Task), hipMemcpyDeviceToHost, streams));
                CUDA_CALL(hipEventRecord(seeding_finished[g_idx][s], streams));
               
#ifdef USE_GPU_DIFFUSE
                CUDA_CALL(hipEventSynchronize((seeding_finished[g_idx][s])));
                CUDA_CALL(hipEventDestroy(seeding_finished[g_idx][s]));

                res_s[g][s].resize(*task_num_host[g_idx][s]);
                vector<int>num_task_vec;
                // printf("NumTask %d\n", *task_num_host[g_idx][s]);
                assert(*task_num_host[g_idx][s] < MaxNumBatch * BatchSize);
                for(size_t cur = 0; cur < MaxNumBatch; cur ++){
                    size_t it = cur * BatchSize;
                    int num_task = BatchSize;
                    if(it > *task_num_host[g_idx][s]) break;
                    if(it+BatchSize >= *task_num_host[g_idx][s]){
                        num_task =  *task_num_host[g_idx][s] - it;
                    }
                    num_task_vec.push_back(num_task);
                    
                    
                    CUDA_CALL(hipMallocAsync((void**)&rd[g_idx][s][cur], direct_matrixSize * BatchSize * sizeof(int), streams));
                    CUDA_CALL(hipMallocAsync((void**)&rt[g_idx][s][cur], MaxBW * (TILE_SIZE + 1) * BatchSize * sizeof(record), streams));
                    CUDA_CALL(hipMemsetAsync(rd[g_idx][s][cur], 0, direct_matrixSize * BatchSize * sizeof(int),  streams));
                    CUDA_CALL(hipMemsetAsync(rt[g_idx][s][cur], 0, MaxBW * (TILE_SIZE + 1) * BatchSize * sizeof(record),  streams));
                    
                    CUDA_CALL(hipMallocAsync((void**)&cigar_op_d[g_idx][s][cur], BatchSize * sizeof(char) * MaxAlignLen, streams));
                    CUDA_CALL(hipMallocAsync((void**)&cigar_cnt_d[g_idx][s][cur], BatchSize * sizeof(int) * MaxAlignLen, streams));

                    CUDA_CALL(hipMallocAsync((void**)&cigar_len_d[g_idx][s][cur], BatchSize * sizeof(int), streams));

                    CUDA_CALL(hipMallocAsync((void**)&score_d[g_idx][s][cur], BatchSize * sizeof(int), streams));
                    
                    CUDA_CALL(hipMallocAsync((void**)&q_end_d[g_idx][s][cur], BatchSize * sizeof(size_t), streams));
                    CUDA_CALL(hipMallocAsync((void**)&s_end_d[g_idx][s][cur], BatchSize * sizeof(size_t), streams));
                    
                    // assert(num_task == num_task_vec[cur]);
                    banded_sw_kernel<<<blocks,threadsPerBlock,0,streams>>>(
                                    num_task_vec[cur],
                                    q_lengths_dev[g_idx], q_offset_dev[g_idx], task_dev[g_idx][s]+it,
                                    query_dev,subj_dev + s_begin_vec[s], s_length[s],
                                    score_d[g_idx][s][cur],
                                    q_end_d[g_idx][s][cur],s_end_d[g_idx][s][cur],
                                    cigar_op_d[g_idx][s][cur],cigar_cnt_d[g_idx][s][cur],cigar_len_d[g_idx][s][cur],
                                    rd[g_idx][s][cur],rt[g_idx][s][cur],band_width,
                                    BLOSUM62_d);
                    CUDA_CALL(hipEventRecord(kernels_done[g_idx][s][cur], streams));                
                    CUDA_CALL(hipStreamWaitEvent(copy_streams[cur], kernels_done[g_idx][s][cur], 0));
                    CUDA_CALL(hipEventDestroy(kernels_done[g_idx][s][cur]));

                    CUDA_CALL(hipMemcpyAsync(score_h[g_idx][s][cur], score_d[g_idx][s][cur], BatchSize * sizeof(int), hipMemcpyDeviceToHost, copy_streams[cur]));
                    CUDA_CALL(hipMemcpyAsync(q_end_h[g_idx][s][cur], q_end_d[g_idx][s][cur], BatchSize * sizeof(size_t), hipMemcpyDeviceToHost,copy_streams[cur]));
                    CUDA_CALL(hipMemcpyAsync(s_end_h[g_idx][s][cur], s_end_d[g_idx][s][cur], BatchSize * sizeof(size_t), hipMemcpyDeviceToHost, copy_streams[cur]));
                    
                    CUDA_CALL(hipMemcpyAsync(cigar_op_h[g_idx][s][cur], cigar_op_d[g_idx][s][cur], BatchSize * sizeof(char) * MaxAlignLen, hipMemcpyDeviceToHost,copy_streams[cur]));
                    CUDA_CALL(hipMemcpyAsync(cigar_cnt_h[g_idx][s][cur], cigar_cnt_d[g_idx][s][cur], BatchSize * sizeof(int) * MaxAlignLen, hipMemcpyDeviceToHost,copy_streams[cur]));
                    CUDA_CALL(hipMemcpyAsync(cigar_len_h[g_idx][s][cur], cigar_len_d[g_idx][s][cur], BatchSize * sizeof(int), hipMemcpyDeviceToHost,copy_streams[cur]));
                    
                    CUDA_CALL(hipEventRecord(copies_done[g_idx][s][cur], copy_streams[cur]));
                    CUDA_CALL(hipFreeAsync(rd[g_idx][s][cur], streams));
                    CUDA_CALL(hipFreeAsync(rt[g_idx][s][cur], streams));
                    CUDA_CALL(hipFreeAsync(score_d[g_idx][s][cur], streams));
                    // CUDA_CALL(hipEventSynchronize((copies_done[g_idx][s][cur])));
                    // for(int i = 0; i < num_task_vec[cur]; ++ i){
                    //     assert(cigar_len_h[i] > 0);
                    // }
                    report_threads[g_idx][s][cur] = thread(call_results,
                                                    ref(copies_done[g_idx][s][cur]),
                                                    ref(streams),
                                                    s_end_d[g_idx][s][cur], q_end_d[g_idx][s][cur],
                                                    cigar_op_d[g_idx][s][cur],  cigar_cnt_d[g_idx][s][cur],  cigar_len_d[g_idx][s][cur],
                                                    query, subj[s],
                                                    task_host[g_idx][s]+it, num_task_vec[cur],
                                                    cigar_len_h[g_idx][s][cur], cigar_op_h[g_idx][s][cur], cigar_cnt_h[g_idx][s][cur],
                                                    q_end_h[g_idx][s][cur], s_end_h[g_idx][s][cur],
                                                    score_h[g_idx][s][cur],
                                                    ref(res_s[g][s]), it,
                                                    pool, ref(rs_report[g_idx][s][cur]));
                    
                    
                }
                // CUDA_CALL(hipDeviceSynchronize());
#else
                result_threads[g_idx][s] = thread(handle_results, ref(seeding_finished[g_idx][s]), query, subj[s], task_host[g_idx][s], task_num_host[g_idx][s], ref(q_groups[g]), s_length[s], s, ref(res_s[g][s]), ref(sw_tasks[g][s]), pool, ref(rs[g_idx][s]));
                cout << "=";
#endif
            }
        }
#ifdef USE_GPU_DIFFUSE
        struct timeval join_start, join_end;
        gettimeofday(&join_start, NULL);
        for (int g = g_begin; g < g_begin + n_groups; g++)
        {
            int g_idx = g - g_begin;
            for(int s = 0; s < NUM_STREAM; ++ s){
                
                for(int cur = 0; cur < MaxNumBatch; ++ cur){
                    size_t it = cur * BatchSize;
                    if(it > *task_num_host[g_idx][s]) break;
                    if (report_threads[g_idx][s][cur].joinable()) {
                        report_threads[g_idx][s][cur].join();
                    }

                    // for (auto &r : rs_report[s][cur]) {
                    //     if (r.valid()) {
                    //         r.get();
                    //     }
                    // }
                }
            }
        }
        gettimeofday(&join_end, NULL);
        cout << "Join Time: " << timeuse(join_start, join_end) << endl;
#endif
        CUDA_CALL(hipDeviceSynchronize());
        nvtxRangePop();
        gettimeofday(&t_end, NULL);
        time_prof.gpu_time += timeuse(t_start, t_end);
        group_time += timeuse(t_start, t_end);
        // cout << "GPU computing Time: " << timeuse(t_start, t_end) << endl;

        if (g_begin == 0)
        {
            gettimeofday(&tt_start, NULL);
            for (int s = 0; s < NUM_STREAM; s++)
            {
                string fname = db_name + "_" + to_string(db_num) + "_" + to_string(s) + ".name";
                int fd = open(fname.data(), O_RDONLY);
                if (fd == -1)
                {
                    std::cerr << "Error opening '" << fname << ". Bailing out." << std::endl;
                    exit(1);
                }
                size_t len = lseek(fd, 0, SEEK_END);
                char *map = (char *)mmap(NULL, len, PROT_READ, MAP_PRIVATE, fd, 0);
                close(fd);
                s_name[s] = (char *)malloc(len);
                memcpy(s_name[s], map, len);
                munmap(map, len);

                s_num[s] = load_offsets(db_name + "_" + to_string(db_num) + "_" + to_string(s), s_offsets[s], sn_offsets[s]);
            }
            gettimeofday(&t_end, NULL);
            time_prof.name_time += timeuse(tt_start, t_end);
            group_time += timeuse(tt_start, t_end);
            // cout << "Load seqs name Time: " << timeuse(tt_start, t_end) << endl;
        }

        gettimeofday(&tt_start, NULL);

        int hsp_count = 0;

        for (int s = 0; s < NUM_STREAM; s++)
        {
            for (int g = g_begin; g < g_begin + n_groups; g++)
            {
                
                int g_idx = g - g_begin;
                // printf("@@ g=%d|s=%d res_s[g][s]%d task_num_host %d\n",g,s,res_s[g][s].size(),*task_num_host[g_idx][s]);
#ifndef USE_GPU_DIFFUSE
                result_threads[g_idx][s].join();
#endif
                hsp_count += res_s[g][s].size();
                cout << "=";
// #ifndef USE_GPU_SW
#ifndef USE_GPU_DIFFUSE
                for (auto &r : rs[g_idx][s])
                    r.get();
#endif
                proceed_result(res, ref(res_s[g][s]), query, subj[s], q_groups[g], s_name[s], s_offsets[s], sn_offsets[s], s_num[s], total_db_size);
                cout << "=";
// #endif
            }
        }

        // g_begin += n_groups;

        gettimeofday(&t_end, NULL);
        time_prof.cpu_time += timeuse(tt_start, t_end);
        group_time += timeuse(tt_start, t_end);
        cout << "] " << group_time << "s, " << hsp_count << " HSPs" << endl;
    }

    gettimeofday(&t_start, NULL);

    n_groups = q_groups.size();

    if (n_groups > MAX_GROUPS_PER_ROUND){
        n_groups = MAX_GROUPS_PER_ROUND;
    }
#ifdef USE_GPU_DIFFUSE
    for(int g = 0; g < n_groups; ++ g){

        for(int s = 0; s < NUM_STREAM; ++ s){
            for(int cur = 0; cur < MaxNumBatch; ++ cur){
                CUDA_CALL(hipHostFree(score_h[g][s][cur])); 
                
                CUDA_CALL(hipHostFree(s_end_h[g][s][cur])); 
                CUDA_CALL(hipHostFree(q_end_h[g][s][cur])); 
                
                CUDA_CALL(hipHostFree(cigar_op_h[g][s][cur])); 
                CUDA_CALL(hipHostFree(cigar_cnt_h[g][s][cur])); 
                CUDA_CALL(hipHostFree(cigar_len_h[g][s][cur])); 
            }
        }
    }
#endif
    // for(int s = 0; s < NUM_STREAM; ++ s){
#ifdef USE_GPU_DIFFUSE
        CUDA_CALL(hipFreeAsync(BLOSUM62_d, malloc_streams)); 
        for(int cur = 0; cur < MaxNumBatch; ++ cur){
            CUDA_CALL(hipStreamDestroy(copy_streams[cur]));
            
        }   
#endif
        CUDA_CALL(hipStreamDestroy(streams));
        CUDA_CALL(hipStreamDestroy(malloc_streams));
    // }

    for (int g = 0; g < n_groups; g++)
    {
        for (int s = 0; s < NUM_STREAM; s++)
        {
            // destroy_hashtable(pHashTable[s]);
            CUDA_CALL(hipFree(task_dev[g][s]));
            CUDA_CALL(hipFree(task_num_dev[g][s]));
        }
        CUDA_CALL(hipFree(q_num_dev[g]));

        CUDA_CALL(hipFree(q_idx_dev[g]));

        CUDA_CALL(hipFree(q_lengths_dev[g]));
        CUDA_CALL(hipFree(q_offset_dev[g]));

        CUDA_CALL(hipFree(index_size_dev[g]));

        CUDA_CALL(hipFree(threshold_dev[g]));
    }

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);

#ifdef USE_GPU_SW

    gettimeofday(&t_start, NULL);
    // char* query_dev;
    // CUDA_CALL(hipMalloc((void **)&query_dev, total_len_query));
    // CUDA_CALL(hipMemcpy(query_dev, query, total_len_query, hipMemcpyHostToDevice));
    // sw_tasks_total.q = query;
    for (int s = 0; s < NUM_STREAM; s++)
    {
        sw_tasks_total.c_all[s] = subj[s];
        sw_tasks_total.c_offs[s] = s==0? 0: sw_tasks_total.c_offs[s-1] +s_length[s-1];
    }
    kernel_run(ref(sw_tasks_total), query_dev, subj_dev, res_s, band_width);
    // gasal_run(sw_tasks_total, res_s, query_dev, subj_dev, q_groups.size(), band_width);
    cout << "Done.\t[";

    gettimeofday(&t_end, NULL);
    time_prof.gpu_time += timeuse(t_start, t_end);
    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipFree(query_dev));
    for (int s = 0; s < NUM_STREAM; s++)
    {
        for (int g = 0; g < q_groups.size(); g++)
        {
            proceed_result(res, res_s[g][s], query, subj[s], q_groups[g], s_name[s], s_offsets[s], sn_offsets[s], s_num[s], total_db_size);
        }
        cout << "=";
    }
    cout << "] ";
    gettimeofday(&t_end, NULL);
    cout << timeuse(t_start, t_end) <<"s" << endl;
    time_prof.cpu_time += timeuse(t_start, t_end);
#endif

    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipFree(subj_dev));

    for (int s = 0; s < NUM_STREAM; s++)
    {
        free(s_name[s]);
        free(sn_offsets[s]);
        free(s_offsets[s]);
    }

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);
}

void blastp(string argv_query, vector<string> argv_dbs, string argv_out)
{
    vector<uint32_t> q_offsets;
    vector<string> q_names;
    char *query;

    struct timeval t_start, t_end;
    gettimeofday(&t_start, NULL);
    uint32_t q_length = load_fasta(argv_query.data(), query, q_offsets, q_names);

    q_offsets.push_back(q_length);

    vector<uint32_t> q_lengths;
    for (int i = 0; i < q_offsets.size() - 1; i++)
    {
        q_lengths.push_back(q_offsets[i + 1] - q_offsets[i] - 1);
    }
    int n_query = q_offsets.size() - 1;
    gettimeofday(&t_end, NULL);
    cout << "Load query Time: " << timeuse(t_start, t_end) << endl;
    gettimeofday(&t_start, NULL);

    vector<SWResult> res_d[n_query];

    size_t max_db_size = 0;
    size_t total_db_size = 0;
    vector<int> db_sizes;
    for (int i = 0; i < argv_dbs.size(); i++)
    {
        db_sizes.push_back(check_db(argv_dbs[i].data(), max_db_size, total_db_size));
        if (db_sizes[i] <= 0)
        {
            cout << "DB " << argv_dbs[i] << " not found!" << endl;
            exit(-1);
        }
    }
    cout << "Max db size = " << max_db_size / (1073741824) << " GB" << endl;
    cout << "Total db size = " << (double)total_db_size / (1073741824) << " GB" << endl;
    total_db_size = (total_db_size * 8) / 5;

    size_t max_hashtable_capacity;
    uint32_t max_n_query;
    vector<QueryGroup> q_groups = init_query_group(n_query, max_db_size, q_lengths, q_offsets, query, max_hashtable_capacity, max_n_query);

    // init_hashtable_capacity(n_query, max_db_size, q_lengths);

    // cout << "Each hash table size = ";
    // for (int i = 0; i < n_query; i++)
    // {
    //     cout << (double)kHashTableCapacity_host[i] * sizeof(KeyValue) * NUM_STREAM / (1073741824) << " ";
    // }
    // cout << "GB, total size = " << (double)kHashTableOffset_host[n_query] * sizeof(KeyValue) * NUM_STREAM / (1073741824) << " GB." << endl;

    uint32_t n_groups = q_groups.size();
    if (n_groups > MAX_GROUPS_PER_ROUND)
        n_groups = MAX_GROUPS_PER_ROUND;
    // KeyValue *hashtable_host[n_groups][NUM_STREAM];
    Task *task_host[n_groups][NUM_STREAM];
    uint32_t *task_num_host[n_groups][NUM_STREAM];

    for (int g = 0; g < n_groups; g++)
    {
        for (int s = 0; s < NUM_STREAM; s++)
        {
            // CUDA_CALL(hipHostMalloc(&hashtable_host[g][s], max_hashtable_capacity * sizeof(KeyValue)));
            CUDA_CALL(hipHostMalloc(&task_host[g][s], MAX_FILTER_TASK * sizeof(Task)));
            CUDA_CALL(hipHostMalloc(&task_num_host[g][s], sizeof(uint32_t)));
        }
    }

    pool = new ThreadPool(num_threads);

    gettimeofday(&t_end, NULL);
    cout << "Prepare Time: " << timeuse(t_start, t_end) << endl;

    TimeProfile time_prof;

    struct timeval c_start, c_end;
    gettimeofday(&c_start, NULL);
    for (int d = 0; d < argv_dbs.size(); d++)
    {
        string db_name(argv_dbs[d]);
        for (int i = 0; i < db_sizes[d]; i++)
        {
            struct timeval start, end;
            gettimeofday(&start, NULL);
            cout << "Search DB " << d + 1 << "/" << argv_dbs.size() << ", Part " << i + 1 << "/" << db_sizes[d] << endl;

            char *subj[NUM_STREAM];
            size_t s_size[NUM_STREAM];
            size_t s_len[NUM_STREAM];

            for (int s = 0; s < NUM_STREAM; s++)
            {
                load_seq(db_name + "_" + to_string(i), s, ref(subj[s]), ref(s_size[s]));
                s_len[s] = (s_size[s] * 8) / 5;
            }

            search_db_batch(query, subj, q_groups, s_len, task_host, task_num_host, max_hashtable_capacity, max_n_query, q_length, db_name, i, res_d, total_db_size, time_prof);

            for (int s = 0; s < NUM_STREAM; s++)
            {
                munmap(subj[s], s_size[s]);
            }
            gettimeofday(&end, NULL);
            cout << "Total Batch Time: " << timeuse(start, end) << endl;
        }
    }
    gettimeofday(&c_end, NULL);
    cout << "Finish searching." << endl;
    cout << "GPU Calculation time:\t" << time_prof.gpu_time << endl;
    cout << "CPU Calculation time:\t" << time_prof.cpu_time << endl;
    cout << "Others time:\t" << time_prof.mem_time << endl;
    cout << "Load seqs name Time:\t" << time_prof.name_time << endl;
    cout << "Total Calculation Time:\t" << timeuse(c_start, c_end) << endl;

    gettimeofday(&t_start, NULL);

    for (int g = 0; g < n_groups; g++)
        for (int s = 0; s < NUM_STREAM; s++)
        {
            CUDA_CALL(hipHostFree(task_host[g][s]));
            CUDA_CALL(hipHostFree(task_num_host[g][s]));
        }

    gettimeofday(&t_end, NULL);
    cout << "Free memory Time:\t" << timeuse(t_start, t_end) << endl;
    gettimeofday(&t_start, NULL);

    for (int i = 0; i < n_query; i++)
    {
        sort_heap(res_d[i].begin(), res_d[i].end(), [&](const SWResult &sw1, const SWResult &sw2)
                  { return (sw1.e_value == sw2.e_value) ? (sw1.score > sw2.score) : (sw1.e_value < sw2.e_value); });
    }

    int outfmt;
    get_arg("outfmt", outfmt, D_OUTFMT);
    switch (outfmt)
    {
    case 0:
        output_result_tabular(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 1:
        output_result_align(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 2:
        output_result_tabular(argv_out, res_d, query, q_offsets, q_names);
        output_result_fa(argv_out + ".fasta", res_d, query, q_offsets, q_names);
        break;
    case 3:
        output_result_cast(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 4:
        output_result_a3m(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 5:
        output_result_reduce(argv_out, res_d, query, q_offsets, q_names);
        break;
    default:
        break;
    }

    free(query);
    delete pool;

    gettimeofday(&t_end, NULL);

    cout << "Output Time:\t" << timeuse(t_start, t_end) << endl;

    cout << "Finished." << endl;
}
