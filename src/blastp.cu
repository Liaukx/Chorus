#include "hip/hip_runtime.h"
#include "blastp.h"
#include <assert.h>
#include <chrono>
#include <algorithm> 

#define PACK_KEY(k) ((k & ~0x7) | 0x3)

ThreadPool *pool;
mutex mu2;

// vector<SWResult> res_s[MAX_GROUPS_PER_ROUND][NUM_STREAM];

__constant__ uint32_t kHashTableCapacity_dev[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];
__constant__ uint32_t kHashTableOffset_dev[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];

__constant__ int SEED_LENGTH;
__constant__ int QIT_WIDTH;
__constant__ uint32_t MASK;

// 32 bit Murmur3 hash
inline __device__ uint32_t my_hash(uint32_t k, uint32_t kHashTableCapacity)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}

__device__ int insert_ot(KeyValue *hashtable, uint32_t kHashTableCapacity, uint32_t key, uint32_t value)
{
    key = PACK_KEY(key);
    uint32_t slot = my_hash(key, kHashTableCapacity);
    uint32_t b_slot = slot;
    while (true)
    {
        uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
        if (prev == kEmpty || prev == key)
        {
            // hashtable[slot].value = value;
            atomicAdd(&hashtable[slot].value, value);
            return 0;
        }
        slot = (slot + 1) & (kHashTableCapacity - 1);
        if (slot == b_slot)
        {
            return -1;
        }
    }
}

__global__ void seeding_kernel(KeyValue *ht, uint32_t *subj, size_t s_length_block, size_t s_length_total, const uint32_t *q_lengths, const int *q_num, const int *q_idx, int n_query, uint8_t *index_size_dev, uint32_t group_id)
{
    size_t s_begin = ((blockIdx.x * blockDim.x + threadIdx.x) * s_length_block) * 32;

    size_t s_len = s_length_block * 32;
    if (s_begin + s_len >= s_length_total - SEED_LENGTH)
        s_len = s_length_total - SEED_LENGTH - s_begin;
    if (s_len <= 0)
        return;

    size_t s_end = s_begin + s_len;

    for (size_t i = s_begin; i < s_end; i++)
    {
        size_t n_bit = i * 5;
        size_t pos = (n_bit >> 5);
        uint32_t mod = n_bit & 31;
        // assert(pos % 4 == 0);
        uint32_t qit_idx = (subj[pos] >> mod) & MASK;
        if (mod > (31 - (5 * SEED_LENGTH)))
        {
            qit_idx |= (subj[pos + 1] << (32 - mod)) & MASK;
        }

        int hit_size = index_size_dev[qit_idx];

        if (hit_size <= 0)
            continue;

        int qit_p = 0;
        for (int j = 0; j < hit_size; j++)
        {
            int pos = qit_idx * QIT_WIDTH + qit_p;
            int q_num_now = q_num[pos];
            int q_idx_now = q_idx[pos];
            if (q_num_now == -1)
            {
                qit_idx += q_idx_now;
                qit_p = 0;
                pos = qit_idx * QIT_WIDTH;
                q_num_now = q_num[pos];
                q_idx_now = q_idx[pos];
            }

            // printf("%d %d\n",q_num[qit_idx*qit_width+qit_p],q_idx[qit_idx*qit_width+qit_p]);
            unsigned int diag = q_lengths[q_num_now] + i - q_idx_now;
            // KeyValue *pHashTable_addr = ot + q_num_now * kHashTableCapacity_dev[q_num_now];
            KeyValue *pHashTable_addr = ht + kHashTableOffset_dev[group_id][q_num_now];
            int err = insert_ot(pHashTable_addr, kHashTableCapacity_dev[group_id][q_num_now], diag, 1);
            // assert(err != -1);
            if (err == -1)
            {
                printf("Voting Hash Table Full! G%uQ%uK%u\n", group_id, q_num_now, kHashTableCapacity_dev[group_id][q_num_now]);
            }
            qit_p++;
        }
    }
}

__global__ void filter_kernel(KeyValue *ht, Task *tasks, uint32_t *num_task, uint32_t *threshold, uint32_t group_id)
{
    uint32_t q_id = blockIdx.x;
    KeyValue *h_begin = ht + kHashTableOffset_dev[group_id][q_id];

    size_t each_length = (kHashTableCapacity_dev[group_id][q_id] - 1) / blockDim.x + 1;
    h_begin += each_length * threadIdx.x;
    KeyValue *h_end = h_begin + each_length;

    KeyValue *total_end = ht + kHashTableOffset_dev[group_id][q_id] + kHashTableCapacity_dev[group_id][q_id];
    h_end = h_end > total_end ? total_end : h_end;

    Task *task_begin = tasks;

    for (KeyValue *kv = h_begin; kv < h_end; kv++)
    {
        if (kv->key != kEmpty && kv->value != kEmpty && kv->value >= threshold[q_id])
        {
            uint32_t idx = atomicAdd(num_task, 1);
            if (idx >= MAX_FILTER_TASK)
            {
                printf("Filter Task Vector Full! G%uQ%uT%u\n", group_id, q_id, idx);
                return;
            }
            task_begin[idx].key = kv->key;
            task_begin[idx].value = kv->value;
            task_begin[idx].q_id = q_id;
        }
    }

    // size_t total_length = kHashTableOffset_dev[group_id][n_query-1] + kHashTableCapacity_dev[group_id][n_query-1];
    // size_t each_length = (total_length-1)/b + 1;
}
// TODO error in Score 
__global__ void banded_sw_kernel(uint32_t* q_lens, uint32_t* q_idxs,
                //  size_t* diags,
                Task* task,
                const char* q, const char* c, size_t c_len,
                int *rd, record* rt_d,int band_width,
                int * score_d,
                // int* q_len_d,int* s_len_d,
                size_t* q_end_d, size_t* s_end_d,
                char* cigar_op_d, int* cigar_cnt_d,int* cigar_len_d,
                int* BLOSUM62_d){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    size_t n = q_lens[task[idx].q_id];
    if(n > MaxQueryLen) printf("## Query Len %ld\n", n);
    assert( n < MaxQueryLen);
   
    size_t q_idx  = q_idxs[task[idx].q_id];
    size_t diag  = task[idx].key;
    
    int64_t c_begin = (int64_t)diag - band_width - n + 2;
    size_t c_end = diag + band_width;
    int* BLOSUM62 = BLOSUM62_d + idx * 26 * 26;
    record* rt = rt_d + idx * MaxBW * (TILE_SIZE + 1);
    
    // __shared__ int shared_BLOSUM62[26 * 26];
    
    // for(size_t i = 0; i < MaxBW; ++ i){
    //     for(size_t j = 0; j < MaxQueryLen+1; ++ j){
    //         rd[BatchSize * (i * (MaxQueryLen+1) + j) + idx] = 0;
    //     }
    // }
    // __syncthreads(); // Wait for the copy to compl .ete


    size_t width = 2 * band_width + 1;
    size_t height = MaxQueryLen + 1;
    assert(width < MaxBW);
    
    size_t t_height = TILE_SIZE + 1;
    
    // record *rt = (record *)malloc(width * t_height * sizeof(record));
    // memset(rt, 0, width * t_height * sizeof(record));

    size_t max_q = 0;
    size_t max_c = 0;
    int score = 0, Score = 0;
    // cal maxScore and it's position
    for (size_t it = 0; it * TILE_SIZE < n; it++) {
        
        size_t q_offset = it * TILE_SIZE;

        for(size_t _q = 0; _q < t_height-1 && q_offset + _q < n; ++_q){
            for(size_t _c = 0; _c < width-2; ++_c){
                
                if(c_begin + _c+ q_offset + _q < 0) continue;
                if(c_begin + _c+ q_offset + _q >= c_len) break;

                char chq = q[q_idx + q_offset + _q];
                char chc = get_char_d(c, c_begin + q_offset + _c + _q);
                
                if (chq == END_SIGNAL || chc == END_SIGNAL)
                {
                    continue;
                }
                //rt(_q,_c) -> (_q+1) * width + _c + 1
                // logical m(_q,_c).x = max(m(_q-1,_c).x + SCORE_GAP_EXT, m(_q-1,_c).m +SCORE_GAP, 0 );
                // logical m(_q,_c).y = max(m(_q,_c-1).y + SCORE_GAP_EXT, m(_q,_c-1).m +SCORE_GAP, 0 );
                // logical m(_q,_c).m = max(m(_q-1,_c-1).y,m(_q-1,_c-1).x,m(_q-1,_c-1).m, 0 );
                
                rt[calIndex(_q,_c,MaxBW)].x = max3(rt[calTop(_q,_c,MaxBW)].x + SCORE_GAP_EXT,  rt[calTop(_q,_c,MaxBW)].m + SCORE_GAP, 0);
                rt[calIndex(_q,_c,MaxBW)].y = max3(rt[calLeft(_q,_c,MaxBW)].y + SCORE_GAP_EXT, rt[calLeft(_q,_c,MaxBW)].m + SCORE_GAP, 0);

                if (chq == ILLEGAL_WORD || chc == ILLEGAL_WORD)
                {
                    // illegal word
                    rt[calIndex(_q,_c,MaxBW)].m = 0;
                }
                else
                {
                    rt[calIndex(_q,_c,MaxBW)].m = max2(max3(rt[calDiag(_q,_c,MaxBW)].x, rt[calDiag(_q,_c,MaxBW)].y, rt[calDiag(_q,_c,MaxBW)].m) + BLOSUM62[chq * 26 + chc], 0);
                }

                score = max3(rt[calIndex(_q,_c,MaxBW)].x, rt[calIndex(_q,_c,MaxBW)].y, rt[calIndex(_q,_c,MaxBW)].m);
                
                // printf("(q = %c,c = %c) BLOSUM62 = %d rt[_q * width + _c].s = %d\n", chq+65,chc+65,BLOSUM62[chq * 26 + chc], rt[_q * width + _c].s);
                // (rd + idx*direct_matrixSize)[_c * height + _q + q_offset] = (score == rt[_q * width + _c].x)*TOP + (score == rt[_q * width + _c].y)*LEFT + (rt[_c * height + _q + q_offset].m)*DIAG; 
                if(score)
                    rd[calIndex(_c, _q+q_offset,height) * BatchSize + idx] = \
                        (score == rt[calIndex(_q,_c,MaxBW)].m) ? DIAG : \
                        ((score == rt[calIndex(_q,_c,MaxBW)].y) ? LEFT :TOP );
                
                if (Score < score)
                {
                    Score = score;
                    max_c = _c;
                    max_q = _q + q_offset;
                }
                // printf("(q = %c,c = %c) score = %d maxScore = %d direction = %d\n", chq+65,chc+65,r[_q*width + _c].s,r[max_c * height + max_q].s,r[_q * width + _c].d);
            }
        }
        memcpy(rt,rt + (t_height - 1) * MaxBW ,MaxBW * sizeof(record));
        // Hit when target is not long enough, there are some cells should be zero
        memset(rt + MaxBW, 0, (t_height - 1) * MaxBW * sizeof(record));

    }

    score_d[idx] = Score;
    // res[idx].score = Score;
    assert(Score != 0);

    size_t cur_q= max_q;
    size_t cur_c = max_c;

    q_end_d[idx] = cur_q + q_idx;
    s_end_d[idx] = c_begin + cur_c + cur_q;

    int cnt_q = 0, cnt_c = 0;
    int cigar_len = 0;
    while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx])
    {
        int d = rd[BatchSize * calIndex(cur_c,cur_q,height) + idx];
        // size_t res_q = (d&0x01) ? (cur_q + q_idx) : (size_t)-1;
        // size_t res_c = (d&0x02) ? (c_begin + cur_c + cur_q) : (size_t)-1;
        
        // q_res_d[idx* MaxAlignLen + (cnt_q)] = (res_q);
        // s_res_d[idx* MaxAlignLen + (cnt_c)] = (res_c);
        int cur_cigar_cnt = 0;
        while (rd[BatchSize * calIndex(cur_c,cur_q,height) + idx] && rd[BatchSize * calIndex(cur_c,cur_q,height) + idx]==d){
            cur_cigar_cnt ++;
            
            //TOP 01b, left 10b, diag 11b
            //DIAG : cur_q -= 1
            //TOP : cur_q -= 1, cur_c += 1;
            //LEFT : cur_c -= 1
            cur_q -= (d == DIAG || d == TOP);
            cur_c += (d == TOP); // Increment cur_c if TOP (01b)
            cur_c -= (d == LEFT); // Decrement cur_c if LEFT (10b)
        }
        (cigar_cnt_d + idx * MaxAlignLen)[cigar_len] = cur_cigar_cnt;
        (cigar_op_d + idx * MaxAlignLen)[cigar_len++] = ((d==DIAG)?'M':((d==TOP)?'D':'I'));
    }

    // free(rt);
    assert(cigar_len > 0);
    cigar_len_d[idx] = cigar_len;
}

void cigar_to_index(int cigar_len, char* cigar_op, int* cigar_cnt,
               size_t q_start,
               size_t c_start,
               vector<size_t>& q_res, vector<size_t>& s_res)
{
    size_t cur_q = q_start;
    size_t cur_c = c_start;
    for(int i = 0; i < cigar_len; i ++){
        int cur = cigar_cnt[i];
        char op = cigar_op[i];
        int d = ((op=='M')?DIAG:((op=='D')?TOP:LEFT));
        

        for(int j = 0; j < cur; ++ j){

            int tmp_q = (d&0x01) ? (cur_q) : -1;
            int tmp_c = (d&0x02) ? (cur_c) : -1;
            q_res.push_back(tmp_q);
            s_res.push_back(tmp_c);

            //TOP 01b, left 10b, diag 11b
            //DIAG : cur_q -= 1, cur_c -= 1
            //TOP : cur_q -= 1, 
            //LEFT : cur_c -= 1
            cur_q -= (d == DIAG || d == TOP);
            cur_c -= (d == LEFT || d == DIAG); // Decrement cur_c if LEFT (10b)
        }
    
    }
    reverse(q_res.begin(),q_res.end());
    reverse(s_res.begin(),s_res.end());
    assert(q_res.size() && s_res.size());
}

#ifdef USE_GPU_SW
void handle_results(hipEvent_t &stream, Task *task_host, uint32_t *num_task, QueryGroup &q_group, size_t s_length, int stream_id, vector<SWResult> &res, SWTasks &sw_task)
{
    hipEventSynchronize(stream);
    mu2.lock();
    cout << "=";
    res.clear();
    size_t n_task_pre = sw_task.num_task;
    size_t n_task = sw_task.num_task + *num_task;
    sw_task.c_len += s_length;
    sw_task.q_idxs.resize(n_task);
    sw_task.q_lens.resize(n_task);
    sw_task.q_len4_offs.resize(n_task+1);
    sw_task.s_len4_offs.resize(n_task+1);
    sw_task.diags.resize(n_task);
    sw_task.info.resize(n_task);
    Task *t_begin = task_host;
    sw_task.num_task = n_task;
    res.resize(*num_task);
#pragma omp parallel for
    for (int i = 0; i < *num_task; i++)
    {
        Task &kv = *(t_begin + i);
        sw_task.q_idxs[i + n_task_pre]=q_group.offset[kv.q_id];
        sw_task.q_lens[i + n_task_pre]=q_group.length[kv.q_id];
        sw_task.diags[i + n_task_pre] =  sw_task.c_offset + kv.key;
        sw_task.info[i+ n_task_pre].group_id = q_group.group_id;
        sw_task.info[i+ n_task_pre].stream_id = stream_id;
        sw_task.info[i+ n_task_pre].idx = i;
        res[i].num_q = kv.q_id;
    }

    for (int i = 0; i < *num_task; i++)
    {
        int q_len4 = sw_task.q_lens[i+ n_task_pre];
        q_len4 = q_len4 % 4? q_len4 + (4 - (q_len4 % 4)) : q_len4;
        int s_len4 = sw_task.q_lens[i+ n_task_pre] + (band_width << 1);
        s_len4 = s_len4 %4? s_len4 + (4-(s_len4%4)):s_len4;
        sw_task.q_len4_offs[i+ n_task_pre+1] = sw_task.q_len4_offs[i+ n_task_pre] + q_len4;
        sw_task.s_len4_offs[i+ n_task_pre+1] = sw_task.s_len4_offs[i+ n_task_pre] + s_len4;
    }

    sw_task.c_offset += s_length;
    mu2.unlock();

}
#else
void handle_results(hipEvent_t &stream, const char *query, const char *subj, Task *task_host, uint32_t *num_task, QueryGroup &q_group, size_t s_length, int stream_id, vector<SWResult> &res, SWTasks &sw_task, ThreadPool *pool, vector<future<int>> &rs)
{
    hipEventSynchronize(stream);
    cout << "=";
    res.clear();
    res.resize(*num_task);
    sw_task.q = query;
    sw_task.c = subj;
    sw_task.c_len = s_length;
    sw_task.q_idxs.resize(*num_task);
    sw_task.q_lens.resize(*num_task);
    sw_task.diags.resize(*num_task);
    Task *t_begin = task_host;
    sw_task.num_task = *num_task;
#pragma omp parallel for
    for (int i = 0; i < *num_task; i++)
    {
        Task &kv = *(t_begin + i);
        sw_task.q_idxs[i]=q_group.offset[kv.q_id];
        sw_task.q_lens[i]=q_group.length[kv.q_id];
        sw_task.diags[i]=kv.key;
        res[i].num_q = kv.q_id;
    }
    mu2.lock();
    for (int i = 0; i < sw_task.num_task; ++i)
    {
        rs.emplace_back(pool->enqueue([&, i]
                                      {
            smith_waterman_kernel(i,&res[i],&sw_task);
            return i; }));
    }
    mu2.unlock();
}
#endif

void search_db_batch(const char *query, char *subj[], vector<QueryGroup> &q_groups, size_t s_length[], Task *task_host[][NUM_STREAM], uint32_t *task_num_host[][NUM_STREAM], size_t max_hashtable_capacity, uint32_t max_n_query, uint32_t total_len_query, string db_name, uint32_t db_num, vector<SWResult> *res, size_t total_db_size, TimeProfile &time_prof)
{
    struct timeval t_start, t_end, tt_start;

    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(SEED_LENGTH), &seed_length, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(QIT_WIDTH), &qit_width, sizeof(int)));
    uint32_t mask = (uint32_t)pow(2, 5 * seed_length) - 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MASK), &mask, sizeof(uint32_t)));

    size_t sum_s_len = 0;
    for (int i = 0; i < NUM_STREAM; i++)
    {
        sum_s_len += s_length[i];
        assert(s_length[i] % 32 == 0);
    }

    char *subj_dev;
    CUDA_CALL(hipMalloc((void **)&subj_dev, sum_s_len / 8 * 5));

    int n_groups = q_groups.size();
    if (n_groups > MAX_GROUPS_PER_ROUND)
        n_groups = MAX_GROUPS_PER_ROUND;

    int *q_num_dev[n_groups];
    int *q_idx_dev[n_groups];
    uint32_t *q_lengths_dev[n_groups];
    uint8_t *index_size_dev[n_groups];
    uint32_t *threshold_dev[n_groups];
    uint32_t *q_offset_dev[n_groups];

    uint32_t kHashTableCapacity_host[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];
    uint32_t kHashTableOffset_host[MAX_GROUPS_PER_ROUND][MAX_QUERY_PER_GROUP];

    for (int g = 0; g < n_groups; g++)
    {
        CUDA_CALL(hipMalloc((void **)&q_num_dev[g], qit_length * qit_width * sizeof(int)));
        CUDA_CALL(hipMalloc((void **)&q_idx_dev[g], qit_length * qit_width * sizeof(int)));
        CUDA_CALL(hipMalloc((void **)&q_lengths_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
        CUDA_CALL(hipMalloc((void **)&q_offset_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
        CUDA_CALL(hipMalloc((void **)&index_size_dev[g], qit_length * sizeof(uint8_t)));
        CUDA_CALL(hipMalloc((void **)&threshold_dev[g], MAX_QUERY_PER_GROUP * sizeof(uint32_t)));
    }
    KeyValue *pHashTable[NUM_STREAM];
    Task *task_dev[NUM_STREAM];
    uint32_t *task_num_dev[NUM_STREAM];

    for (int s = 0; s < NUM_STREAM; s++)
    {
        // pHashTable[s] = create_hashtable(max_hashtable_capacity);
        CUDA_CALL(hipMalloc((void **)&task_dev[s], MAX_FILTER_TASK * sizeof(Task)));
        CUDA_CALL(hipMemset(task_dev[s], 0, MAX_FILTER_TASK * sizeof(Task)));
        CUDA_CALL(hipMalloc((void **)&task_num_dev[s], sizeof(uint32_t)));
        CUDA_CALL(hipMemset(task_num_dev[s], 0, sizeof(uint32_t)));
    }

    char *s_name[NUM_STREAM] = {0};
    size_t *s_offsets[NUM_STREAM] = {0};
    size_t *sn_offsets[NUM_STREAM] = {0};
    size_t s_num[NUM_STREAM] = {0};

    int mingridsize_seeding, mingridsize_filter;
    int threadblocksize_seeding, threadblocksize_filter;
    CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&mingridsize_seeding, &threadblocksize_seeding, seeding_kernel, 0, 0));
    CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&mingridsize_filter, &threadblocksize_filter, filter_kernel, 0, 0));

    // cout << "Seeding Block size:" << threadblocksize_seeding <<"," << mingridsize_seeding <<endl;
    // cout << "Filter Block size:" << threadblocksize_filter <<"," << mingridsize_filter <<endl;

    size_t free_byte, total_byte;
    CUDA_CALL(hipMemGetInfo(&free_byte, &total_byte));
    cout << "GPU mem: " << (double)(total_byte - free_byte) / (1073741824) << " GB / " << (double)total_byte / (1073741824) << " GB" << endl;

#ifndef USE_GPU_SW
    SWTasks sw_tasks[q_groups.size()][NUM_STREAM];
#endif
    SWTasks sw_tasks_total;
    vector<SWResult> res_s[q_groups.size()][NUM_STREAM];

    char* query_dev;
    CUDA_CALL(hipMalloc((void **)&query_dev, total_len_query));
    CUDA_CALL(hipMemcpy(query_dev, query, total_len_query, hipMemcpyHostToDevice));
#ifdef USE_GPU_SW
    sw_tasks_total.q = query;
#endif

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);

    int g_begin = 0;
    while (g_begin < q_groups.size())
    {
        sw_tasks_total.c_offset = 0;
        double group_time = 0;
        cout << "Group " << g_begin + 1 << "/" << q_groups.size() << "\t[";
        gettimeofday(&t_start, NULL);
        n_groups = q_groups.size() - g_begin;
        if (n_groups > MAX_GROUPS_PER_ROUND)
            n_groups = MAX_GROUPS_PER_ROUND;
        for (int g = g_begin; g < g_begin + n_groups; g++)
        {
            int g_idx = g - g_begin;
            CUDA_CALL(hipMemcpy(q_num_dev[g_idx], q_groups[g].qit.q_num, qit_length * qit_width * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_idx_dev[g_idx], q_groups[g].qit.q_idx, qit_length * qit_width * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(index_size_dev[g_idx], q_groups[g].qit.index_size, qit_length * sizeof(uint8_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_lengths_dev[g_idx], q_groups[g].length, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(q_offset_dev[g_idx], q_groups[g].offset, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(threshold_dev[g_idx], q_groups[g].min_diag_hit, MAX_QUERY_PER_GROUP * sizeof(uint32_t), hipMemcpyHostToDevice));
            memcpy(kHashTableCapacity_host[g_idx], q_groups[g].hashtable_capacity, sizeof(uint32_t) * MAX_QUERY_PER_GROUP);
            memcpy(kHashTableOffset_host[g_idx], q_groups[g].hashtable_offset, sizeof(uint32_t) * MAX_QUERY_PER_GROUP);
        }

        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(kHashTableCapacity_dev), kHashTableCapacity_host, sizeof(uint32_t) * MAX_QUERY_PER_GROUP * MAX_GROUPS_PER_ROUND));
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(kHashTableOffset_dev), kHashTableOffset_host, sizeof(uint32_t) * MAX_QUERY_PER_GROUP * MAX_GROUPS_PER_ROUND));

        hipStream_t streams[NUM_STREAM];
        thread result_threads[n_groups][NUM_STREAM];

        hipEvent_t seeding_finished[n_groups][NUM_STREAM];


#ifdef USE_GUP_DIFFUSE
        hipEvent_t sw_kernel_finished[NUM_STREAM][200];

        int direct_matrixSize = (MaxQueryLen+1) * MaxBW;
        int threadsPerBlock = 256;  // 根据 shared memory 限制调整
        int blocks = (BatchSize + threadsPerBlock - 1) / threadsPerBlock;
        
        int* rd[NUM_STREAM];   // direct_matrixSize * BatchSize * sizeof(int)
        record* rt[NUM_STREAM];
        static int* BLOSUM62_d[NUM_STREAM];
        int* score_d[NUM_STREAM][200], *score_h[NUM_STREAM][200];
        
        size_t* q_end_d[NUM_STREAM][200], *q_end_h[NUM_STREAM][200];
        size_t* s_end_d[NUM_STREAM][200], *s_end_h[NUM_STREAM][200];
        char* cigar_op_d[NUM_STREAM][200], *cigar_op_h[NUM_STREAM][200];
        int *cigar_cnt_d[NUM_STREAM][200], *cigar_cnt_h[NUM_STREAM][200];
        int* cigar_len_d[NUM_STREAM][200], *cigar_len_h[NUM_STREAM][200];


        for (int s = 0; s < NUM_STREAM; s++)
        {
            for(int i = 0; i < 200; ++ i){
                score_h[s][i] = (int*) malloc(sizeof(int) * BatchSize);

                q_end_h[s][i] = (size_t*) malloc(sizeof(size_t) * BatchSize);
                s_end_h[s][i] = (size_t*) malloc(sizeof(size_t) * BatchSize);
                cigar_cnt_h[s][i] = (int*) malloc(sizeof(int) * MaxAlignLen * BatchSize);
                cigar_op_h[s][i] = (char*) malloc(sizeof(char) * MaxAlignLen * BatchSize);
                cigar_len_h[s][i] = (int*) malloc(sizeof(int) * BatchSize);
            }
        }
        
        // SWResult_d * res_d[NUM_STREAM], *res_h[NUM_STREAM];  // MaxAlignLen * BatchSize * sizeof(SWResult_d)
        // static int* BLOSUM62_d[NUM_STREAM];
        // for (int s = 0; s < NUM_STREAM; s++)
        // {
        //     res_h[s] = (SWResult_d*) malloc(BatchSize * sizeof(SWResult_d));
        // }
    
#endif

#ifndef USE_GPU_SW
        vector<future<int>> rs[n_groups][NUM_STREAM];
#endif
        size_t s_begin = 0;

        gettimeofday(&t_end, NULL);
        group_time += timeuse(t_start, t_end);
        time_prof.mem_time += timeuse(t_start, t_end);
        // cout << "Prepare mem and data Time: " << timeuse(t_start, t_end) << endl;

        gettimeofday(&t_start, NULL);

        for (int s = 0; s < NUM_STREAM; s++)
        {
            CUDA_CALL(hipStreamCreate(&streams[s]));
#ifdef USE_GUP_DIFFUSE
            CUDA_CALL(hipMallocAsync((void**)&rd[s], direct_matrixSize * BatchSize * sizeof(int),streams[s]));
            CUDA_CALL(hipMallocAsync((void**)&rt[s], MaxBW * (TILE_SIZE + 1) * BatchSize * sizeof(record),streams[s]));
            
            // CUDA_CALL(hipMallocAsync((void**)&res_d[s],BatchSize * sizeof(SWResult_d),streams[s]));
            for(int i = 0; i < 200; ++ i){

                CUDA_CALL(hipMallocAsync((void**)&score_d[s][i],BatchSize * sizeof(int),streams[s]));
                
                CUDA_CALL(hipMallocAsync((void**)&q_end_d[s][i],BatchSize * sizeof(size_t),streams[s]));
                CUDA_CALL(hipMallocAsync((void**)&s_end_d[s][i],BatchSize * sizeof(size_t),streams[s]));
                CUDA_CALL(hipMallocAsync((void**)&cigar_op_d[s][i],BatchSize * sizeof(char) * MaxAlignLen,streams[s]));
                CUDA_CALL(hipMallocAsync((void**)&cigar_cnt_d[s][i],BatchSize * sizeof(int) * MaxAlignLen,streams[s]));
                CUDA_CALL(hipMallocAsync((void**)&cigar_len_d[s][i],BatchSize * sizeof(int),streams[s]));
            }
            
            CUDA_CALL(hipMallocAsync((void**)&BLOSUM62_d[s], 26 * 26 * sizeof(int) * BatchSize,streams[s]));
            for(int i = 0; i < BatchSize; ++ i)
                CUDA_CALL(hipMemcpyAsync(BLOSUM62_d[s] + i * 26 * 26, BLOSUM62, 26 * 26 * sizeof(int),hipMemcpyHostToDevice,streams[s]));
#endif
            pHashTable[s] = create_hashtable_async(max_hashtable_capacity,streams[s]);
            // printf("start stream %d\n", s);
            size_t s_length_stream = s_length[s];
            size_t s_length_stream_byte = s_length_stream / 8 * 5;
            size_t s_length_stream_block = s_length_stream / 32 * 5;
            size_t each_length_block = (s_length_stream_block - 1) / (mingridsize_seeding * threadblocksize_seeding) + 1;

            if (g_begin == 0)
            {
                CUDA_CALL(hipMemcpyAsync(subj_dev + s_begin, subj[s], s_length_stream_byte, hipMemcpyHostToDevice, streams[s]));
            }
            if (STREAM_SYNC && s > 0)
            {
                CUDA_CALL(hipStreamSynchronize(streams[s - 1]));
            }
            for (int g = g_begin; g < g_begin + n_groups; g++)
            {
                int g_idx = g - g_begin;
                CUDA_CALL(hipEventCreate(&seeding_finished[g_idx][s]));
                int n_query = q_groups[g].n_query;
                if (g > 0)
                {
                    CUDA_CALL(hipMemsetAsync(task_dev[s], 0, MAX_FILTER_TASK * sizeof(Task), streams[s]));
                    CUDA_CALL(hipMemsetAsync(task_num_dev[s], 0, sizeof(uint32_t), streams[s]));
                    CUDA_CALL(hipMemsetAsync(pHashTable[s], 0xff, max_hashtable_capacity * sizeof(KeyValue), streams[s]));
                }
                seeding_kernel<<<mingridsize_seeding, threadblocksize_seeding, 0, streams[s]>>>(pHashTable[s], (uint32_t *)(subj_dev + s_begin), each_length_block, s_length_stream, q_lengths_dev[g_idx], q_num_dev[g_idx], q_idx_dev[g_idx], n_query, index_size_dev[g_idx], g_idx);
                filter_kernel<<<n_query, threadblocksize_filter, 0, streams[s]>>>(pHashTable[s], task_dev[s], task_num_dev[s], threshold_dev[g_idx], g_idx);
                // CUDA_CALL(hipMemcpyAsync(hashtable_host[g_idx][s], pHashTable[s], max_hashtable_capacity * sizeof(KeyValue), hipMemcpyDeviceToHost, streams[s]));
                CUDA_CALL(hipMemcpyAsync(task_host[g_idx][s], task_dev[s], MAX_FILTER_TASK * sizeof(Task), hipMemcpyDeviceToHost, streams[s]));
                CUDA_CALL(hipMemcpyAsync(task_num_host[g_idx][s], task_num_dev[s], sizeof(uint32_t), hipMemcpyDeviceToHost, streams[s]));
                CUDA_CALL(hipEventRecord(seeding_finished[g_idx][s]));
// #ifdef USE_GPU_SW
//                 result_threads[g_idx][s] = thread(handle_results, ref(seeding_finished[g_idx][s]), task_host[g_idx][s], task_num_host[g_idx][s], ref(q_groups[g]), s_length[s], s, ref(res_s[g][s]), ref(sw_tasks_total));
#ifdef USE_GUP_DIFFUSE

                hipEventSynchronize(seeding_finished[g_idx][s]);
                size_t cpu_start = 0, n = *task_num_host[g_idx][s];
                // printf("@@ g=%d, s=%d %d\n",g,s,n);
                Task *t_begin = task_host[g_idx][s];
                
                res_s[g][s].resize(n);
                
                for(int it = 0; it < n; ++it) res_s[g][s][it].num_q = task_host[g_idx][s][it].q_id;
                
                // printf("cuda kernel begin\n");
                for(size_t it = 0; it < n; it += BatchSize){
                    if(it+BatchSize >= n){
                        cpu_start = it;
                        break;
                    }
                    CUDA_CALL(hipMemsetAsync(rd[s], 0, direct_matrixSize * BatchSize * sizeof(int), streams[s]));
                    CUDA_CALL(hipMemsetAsync(rt[s], 0, MaxBW * (TILE_SIZE + 1) * BatchSize * sizeof(record), streams[s]));

                    CUDA_CALL(hipEventCreate(&sw_kernel_finished[s][it/BatchSize]));    
                    banded_sw_kernel<<<blocks,threadsPerBlock,0,streams[s]>>>(
                                    q_lengths_dev[g_idx], q_offset_dev[g_idx],task_dev[s]+it,
                                    query_dev,subj_dev + s_begin,s_length[s],
                                    rd[s],rt[s],band_width,
                                    score_d[s][it/BatchSize],
                                    q_end_d[s][it/BatchSize],s_end_d[s][it/BatchSize],
                                    cigar_op_d[s][it/BatchSize],cigar_cnt_d[s][it/BatchSize],cigar_len_d[s][it/BatchSize],
                                    BLOSUM62_d[s]);
                    // CUDA_CALL(hipMemcpyAsync(res_h[s], res_d[s], BatchSize * sizeof(SWResult_d), hipMemcpyDeviceToHost,streams[s]));
                    CUDA_CALL(hipMemcpyAsync(score_h[s][it/BatchSize], score_d[s][it/BatchSize], BatchSize * sizeof(int), hipMemcpyDeviceToHost,streams[s]));
                    // CUDA_CALL(hipMemcpyAsync(q_len_h[s][it/BatchSize], q_len_d[s][it/BatchSize], BatchSize * sizeof(int), hipMemcpyDeviceToHost,streams[s]));
                    // CUDA_CALL(hipMemcpyAsync(s_len_h[s][it/BatchSize], s_len_d[s][it/BatchSize], BatchSize * sizeof(int), hipMemcpyDeviceToHost,streams[s]));
                    // CUDA_CALL(hipMemcpyAsync(q_res_h[s][it/BatchSize], q_res_d[s][it/BatchSize], BatchSize * sizeof(size_t) * MaxAlignLen, hipMemcpyDeviceToHost,streams[s]));
                    // CUDA_CALL(hipMemcpyAsync(s_res_h[s][it/BatchSize], s_res_d[s][it/BatchSize], BatchSize * sizeof(size_t) * MaxAlignLen, hipMemcpyDeviceToHost,streams[s]));
                    
                    CUDA_CALL(hipMemcpyAsync(q_end_h[s][it/BatchSize], q_end_d[s][it/BatchSize], BatchSize * sizeof(size_t), hipMemcpyDeviceToHost,streams[s]));
                    CUDA_CALL(hipMemcpyAsync(s_end_h[s][it/BatchSize], s_end_d[s][it/BatchSize], BatchSize * sizeof(size_t), hipMemcpyDeviceToHost,streams[s]));
                    
                    CUDA_CALL(hipMemcpyAsync(cigar_op_h[s][it/BatchSize], cigar_op_d[s][it/BatchSize], BatchSize * sizeof(char) * MaxAlignLen, hipMemcpyDeviceToHost,streams[s]));
                    CUDA_CALL(hipMemcpyAsync(cigar_cnt_h[s][it/BatchSize], cigar_cnt_d[s][it/BatchSize], BatchSize * sizeof(int) * MaxAlignLen, hipMemcpyDeviceToHost,streams[s]));
                    CUDA_CALL(hipMemcpyAsync(cigar_len_h[s][it/BatchSize], cigar_len_d[s][it/BatchSize], BatchSize * sizeof(int), hipMemcpyDeviceToHost,streams[s]));

                    // CUDA_CALL(hipStreamSynchronize(streams[s]));
                    CUDA_CALL(hipEventRecord(sw_kernel_finished[s][it/BatchSize]));
                    hipEventSynchronize(sw_kernel_finished[s][it/BatchSize]);
                    for (size_t i = 0; i < BatchSize; ++i) {

                        if(cigar_len_h[s][it/BatchSize][i] >= MaxAlignLen || cigar_len_h[s][it/BatchSize][i] < 0){
                            printf("## cigar_len: %d\n",cigar_len_h[s][it/BatchSize][i]);
                        }
                        // assert(cigar_len_h[i] < MaxAlignLen && cigar_len_h[i] < MaxAlignLen);
                        //TODO From cigar to index
                        cigar_to_index(cigar_len_h[s][it/BatchSize][i],cigar_op_h[s][it/BatchSize] + i * MaxAlignLen,cigar_cnt_h[s][it/BatchSize] + i * MaxAlignLen,
                                        q_end_h[s][it/BatchSize][i],s_end_h[s][it/BatchSize][i],
                                        res_s[g][s][it + i].q_res,res_s[g][s][it + i].s_res);
                        
                        res_s[g][s][it + i].score = score_h[s][it/BatchSize][i];
                        
                        // check
                        // SWResult sw_tmp;
                        // cpu_kernel(&sw_tmp,query,subj[s],s_length[s],\
                        //             q_groups[g].offset[task_host[g_idx][s][it + i].q_id],\
                        //             q_groups[g].length[task_host[g_idx][s][it + i].q_id],\
                        //             task_host[g_idx][s][it+i].key,band_width);
                        // assert(sw_tmp.q_res.size() == res_s[g][s][it + i].q_res.size() );             
                        // assert(sw_tmp.s_res.size() == res_s[g][s][it + i].s_res.size() );
                        // for(int cnt = 0; cnt < sw_tmp.q_res.size(); ++ cnt){
                        //     if(!(sw_tmp.q_res[cnt] == res_s[g][s][it + i].q_res[cnt])){
                        //         printf("## error %d\n", cnt);
                        //     }
                        //     assert(sw_tmp.q_res[cnt] == res_s[g][s][it + i].q_res[cnt]);
                        // }
                        // for(int cnt = 0; cnt < sw_tmp.s_res.size(); ++ cnt){
                        //     if(!(sw_tmp.s_res[cnt] == res_s[g][s][it + i].s_res[cnt])){
                        //         printf("## error %d\n", cnt);
                        //     }
                        //     assert(sw_tmp.s_res[cnt] == res_s[g][s][it + i].s_res[cnt]);
                        // }            
                        // assert(sw_tmp.score == res_s[g][s][it + i].score );             
                    }
                    // printf("done : %ld\n", it);
                }
                // printf("cuda kernel finished \n");
                for(size_t it=cpu_start; it < n; it ++){   
                    cpu_kernel(&res_s[g][s][it],query,subj[s],s_length[s],q_groups[g].offset[task_host[g_idx][s][it].q_id],q_groups[g].length[task_host[g_idx][s][it].q_id],task_host[g_idx][s][it].key,band_width);
                    generate_report(&res_s[g][s][it],query, subj[s]);
                }
                // printf("kernel finished \n");
                for(size_t it = 0; it < cpu_start; it ++){
                    generate_report(&res_s[g][s][it],query, subj[s]);
                    // printf("generate done : %ld\n", it);
                }

                    // banded_sw_kernel<<<blocks,threadsPerBlock,0,s>>>(q_lengths_dev[g_idx], q_idx_dev[g_idx],diags_d,query_dev,subj_dev,tasks.c_len,rd,band_width,res_d,BLOSUM62_d);
        
#else
                result_threads[g_idx][s] = thread(handle_results, ref(seeding_finished[g_idx][s]), query, subj[s], task_host[g_idx][s], task_num_host[g_idx][s], ref(q_groups[g]), s_length[s], s, ref(res_s[g][s]), ref(sw_tasks[g][s]), pool, ref(rs[g_idx][s]));
#endif 
            }
            // CUDA_CALL(hipStreamSynchronize(streams[s]));
            destroy_hashtable_async(pHashTable[s],streams[s]);
            s_begin += s_length_stream_byte;
            cout << "=";
        }

        CUDA_CALL(hipDeviceSynchronize());

        gettimeofday(&t_end, NULL);
        time_prof.gpu_time += timeuse(t_start, t_end);
        group_time += timeuse(t_start, t_end);
        // cout << "GPU computing Time: " << timeuse(t_start, t_end) << endl;

        if (g_begin == 0)
        {
            gettimeofday(&tt_start, NULL);
            for (int s = 0; s < NUM_STREAM; s++)
            {
                string fname = db_name + "_" + to_string(db_num) + "_" + to_string(s) + ".name";
                int fd = open(fname.data(), O_RDONLY);
                if (fd == -1)
                {
                    std::cerr << "Error opening '" << fname << ". Bailing out." << std::endl;
                    exit(1);
                }
                size_t len = lseek(fd, 0, SEEK_END);
                char *map = (char *)mmap(NULL, len, PROT_READ, MAP_PRIVATE, fd, 0);
                close(fd);
                s_name[s] = (char *)malloc(len);
                memcpy(s_name[s], map, len);
                munmap(map, len);

                s_num[s] = load_offsets(db_name + "_" + to_string(db_num) + "_" + to_string(s), s_offsets[s], sn_offsets[s]);
            }
            gettimeofday(&t_end, NULL);
            time_prof.name_time += timeuse(tt_start, t_end);
            group_time += timeuse(tt_start, t_end);
            // cout << "Load seqs name Time: " << timeuse(tt_start, t_end) << endl;
        }

        gettimeofday(&tt_start, NULL);

        for (int s = 0; s < NUM_STREAM; s++)
        {
            #ifdef USE_GUP_DIFFUSE
                CUDA_CALL(hipFreeAsync(rd[s], streams[s]));
                CUDA_CALL(hipFreeAsync(rt[s], streams[s]));
                CUDA_CALL(hipFreeAsync(BLOSUM62_d[s], streams[s]));
                CUDA_CALL(hipStreamSynchronize(streams[s]));
                
                for(int i = 0; i < 200; ++ i){
                    
                    CUDA_CALL(hipFreeAsync(score_d[s][i], streams[s]));
                    CUDA_CALL(hipFreeAsync(s_end_d[s][i], streams[s]));
                    CUDA_CALL(hipFreeAsync(q_end_d[s][i], streams[s]));
                    CUDA_CALL(hipFreeAsync(cigar_op_d[s][i], streams[s]));
                    CUDA_CALL(hipFreeAsync(cigar_cnt_d[s][i], streams[s]));
                    CUDA_CALL(hipFreeAsync(cigar_len_d[s][i], streams[s]));

                    
                    free(score_h[s][i]);
                    
                    free(s_end_h[s][i]);
                    free(q_end_h[s][i]);
                    free(cigar_op_h[s][i]);
                    free(cigar_cnt_h[s][i]);
                    free(cigar_len_h[s][i]);
                }
            #endif
            CUDA_CALL(hipStreamDestroy(streams[s]));
        }

        int hsp_count = 0;

        for (int s = 0; s < NUM_STREAM; s++)
        {
            for (int g = g_begin; g < g_begin + n_groups; g++)
            {
                int g_idx = g - g_begin;
#ifndef USE_GUP_DIFFUSE
                result_threads[g_idx][s].join();
#endif
                CUDA_CALL(hipEventDestroy(seeding_finished[g_idx][s]));
                hsp_count += res_s[g][s].size();
                cout << "=";
#ifndef USE_GPU_SW
                for (auto &r : rs[g_idx][s])
                    r.get();
                proceed_result(res, res_s[g][s], query, subj[s], q_groups[g], s_name[s], s_offsets[s], sn_offsets[s], s_num[s], total_db_size);
                cout << "=";
#endif
            }
        }

        g_begin += MAX_GROUPS_PER_ROUND;

        gettimeofday(&t_end, NULL);
        time_prof.cpu_time += timeuse(tt_start, t_end);
        group_time += timeuse(tt_start, t_end);
        cout << "] " << group_time << "s, " << hsp_count << " HSPs" << endl;
    }

    gettimeofday(&t_start, NULL);

    n_groups = q_groups.size();

    if (n_groups > MAX_GROUPS_PER_ROUND)
        n_groups = MAX_GROUPS_PER_ROUND;

    for (int s = 0; s < NUM_STREAM; s++)
    {
        // destroy_hashtable(pHashTable[s]);
        CUDA_CALL(hipFree(task_dev[s]));
        CUDA_CALL(hipFree(task_num_dev[s]));
    }

    for (int g = 0; g < n_groups; g++)
    {
        CUDA_CALL(hipFree(q_num_dev[g]));

        CUDA_CALL(hipFree(q_idx_dev[g]));

        CUDA_CALL(hipFree(q_lengths_dev[g]));
        CUDA_CALL(hipFree(q_offset_dev[g]));

        CUDA_CALL(hipFree(index_size_dev[g]));

        CUDA_CALL(hipFree(threshold_dev[g]));
    }

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);

#ifdef USE_GPU_SW

    gettimeofday(&t_start, NULL);
    // char* query_dev;
    // CUDA_CALL(hipMalloc((void **)&query_dev, total_len_query));
    // CUDA_CALL(hipMemcpy(query_dev, query, total_len_query, hipMemcpyHostToDevice));
    // sw_tasks_total.q = query;
    for (int s = 0; s < NUM_STREAM; s++)
    {
        sw_tasks_total.c_all[s] = subj[s];
        sw_tasks_total.c_offs[s] = s==0? 0: sw_tasks_total.c_offs[s-1] +s_length[s-1];
    }
    kernel_run(ref(sw_tasks_total), query_dev, subj_dev, res_s, band_width);
    // gasal_run(sw_tasks_total, res_s, query_dev, subj_dev, q_groups.size(), band_width);
    cout << "Done.\t[";

    gettimeofday(&t_end, NULL);
    time_prof.gpu_time += timeuse(t_start, t_end);
    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipFree(query_dev));
    for (int s = 0; s < NUM_STREAM; s++)
    {
        for (int g = 0; g < q_groups.size(); g++)
        {
            proceed_result(res, res_s[g][s], query, subj[s], q_groups[g], s_name[s], s_offsets[s], sn_offsets[s], s_num[s], total_db_size);
        }
        cout << "=";
    }
    cout << "] ";
    gettimeofday(&t_end, NULL);
    cout << timeuse(t_start, t_end) <<"s" << endl;
    time_prof.cpu_time += timeuse(t_start, t_end);
#endif

    gettimeofday(&t_start, NULL);

    CUDA_CALL(hipFree(subj_dev));

    for (int s = 0; s < NUM_STREAM; s++)
    {
        free(s_name[s]);
        free(sn_offsets[s]);
        free(s_offsets[s]);
    }

    gettimeofday(&t_end, NULL);
    time_prof.mem_time += timeuse(t_start, t_end);
}

void blastp(string argv_query, vector<string> argv_dbs, string argv_out)
{
    vector<uint32_t> q_offsets;
    vector<string> q_names;
    char *query;

    struct timeval t_start, t_end;
    gettimeofday(&t_start, NULL);
    uint32_t q_length = load_fasta(argv_query.data(), query, q_offsets, q_names);

    q_offsets.push_back(q_length);

    vector<uint32_t> q_lengths;
    for (int i = 0; i < q_offsets.size() - 1; i++)
    {
        q_lengths.push_back(q_offsets[i + 1] - q_offsets[i] - 1);
    }
    int n_query = q_offsets.size() - 1;
    gettimeofday(&t_end, NULL);
    cout << "Load query Time: " << timeuse(t_start, t_end) << endl;
    gettimeofday(&t_start, NULL);

    vector<SWResult> res_d[n_query];

    size_t max_db_size = 0;
    size_t total_db_size = 0;
    vector<int> db_sizes;
    for (int i = 0; i < argv_dbs.size(); i++)
    {
        db_sizes.push_back(check_db(argv_dbs[i].data(), max_db_size, total_db_size));
        if (db_sizes[i] <= 0)
        {
            cout << "DB " << argv_dbs[i] << " not found!" << endl;
            exit(-1);
        }
    }
    cout << "Max db size = " << max_db_size / (1073741824) << " GB" << endl;
    cout << "Total db size = " << (double)total_db_size / (1073741824) << " GB" << endl;
    total_db_size = (total_db_size * 8) / 5;

    size_t max_hashtable_capacity;
    uint32_t max_n_query;
    vector<QueryGroup> q_groups = init_query_group(n_query, max_db_size, q_lengths, q_offsets, query, max_hashtable_capacity, max_n_query);

    // init_hashtable_capacity(n_query, max_db_size, q_lengths);

    // cout << "Each hash table size = ";
    // for (int i = 0; i < n_query; i++)
    // {
    //     cout << (double)kHashTableCapacity_host[i] * sizeof(KeyValue) * NUM_STREAM / (1073741824) << " ";
    // }
    // cout << "GB, total size = " << (double)kHashTableOffset_host[n_query] * sizeof(KeyValue) * NUM_STREAM / (1073741824) << " GB." << endl;

    uint32_t n_groups = q_groups.size();
    if (n_groups > MAX_GROUPS_PER_ROUND)
        n_groups = MAX_GROUPS_PER_ROUND;
    // KeyValue *hashtable_host[n_groups][NUM_STREAM];
    Task *task_host[n_groups][NUM_STREAM];
    uint32_t *task_num_host[n_groups][NUM_STREAM];

    for (int g = 0; g < n_groups; g++)
    {
        for (int s = 0; s < NUM_STREAM; s++)
        {
            // CUDA_CALL(hipHostMalloc(&hashtable_host[g][s], max_hashtable_capacity * sizeof(KeyValue)));
            CUDA_CALL(hipHostMalloc(&task_host[g][s], MAX_FILTER_TASK * sizeof(Task)));
            CUDA_CALL(hipHostMalloc(&task_num_host[g][s], sizeof(uint32_t)));
        }
    }

    pool = new ThreadPool(num_threads);

    gettimeofday(&t_end, NULL);
    cout << "Prepare Time: " << timeuse(t_start, t_end) << endl;

    TimeProfile time_prof;

    struct timeval c_start, c_end;
    gettimeofday(&c_start, NULL);
    for (int d = 0; d < argv_dbs.size(); d++)
    {
        string db_name(argv_dbs[d]);
        for (int i = 0; i < db_sizes[d]; i++)
        {
            struct timeval start, end;
            gettimeofday(&start, NULL);
            cout << "Search DB " << d + 1 << "/" << argv_dbs.size() << ", Part " << i + 1 << "/" << db_sizes[d] << endl;

            char *subj[NUM_STREAM];
            size_t s_size[NUM_STREAM];
            size_t s_len[NUM_STREAM];

            for (int s = 0; s < NUM_STREAM; s++)
            {
                load_seq(db_name + "_" + to_string(i), s, ref(subj[s]), ref(s_size[s]));
                s_len[s] = (s_size[s] * 8) / 5;
            }

            search_db_batch(query, subj, q_groups, s_len, task_host, task_num_host, max_hashtable_capacity, max_n_query, q_length, db_name, i, res_d, total_db_size, time_prof);

            for (int s = 0; s < NUM_STREAM; s++)
            {
                munmap(subj[s], s_size[s]);
            }
            gettimeofday(&end, NULL);
            cout << "Total Batch Time: " << timeuse(start, end) << endl;
        }
    }
    gettimeofday(&c_end, NULL);
    cout << "Finish searching." << endl;
    cout << "GPU Calculation time:\t" << time_prof.gpu_time << endl;
    cout << "CPU Calculation time:\t" << time_prof.cpu_time << endl;
    cout << "Others time:\t" << time_prof.mem_time << endl;
    cout << "Load seqs name Time:\t" << time_prof.name_time << endl;
    cout << "Total Calculation Time:\t" << timeuse(c_start, c_end) << endl;

    gettimeofday(&t_start, NULL);

    for (int g = 0; g < n_groups; g++)
        for (int s = 0; s < NUM_STREAM; s++)
        {
            CUDA_CALL(hipHostFree(task_host[g][s]));
            CUDA_CALL(hipHostFree(task_num_host[g][s]));
        }

    gettimeofday(&t_end, NULL);
    cout << "Free memory Time:\t" << timeuse(t_start, t_end) << endl;
    gettimeofday(&t_start, NULL);

    for (int i = 0; i < n_query; i++)
    {
        sort_heap(res_d[i].begin(), res_d[i].end(), [&](const SWResult &sw1, const SWResult &sw2)
                  { return (sw1.e_value == sw2.e_value) ? (sw1.score > sw2.score) : (sw1.e_value < sw2.e_value); });
    }

    int outfmt;
    get_arg("outfmt", outfmt, D_OUTFMT);
    switch (outfmt)
    {
    case 0:
        output_result_tabular(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 1:
        output_result_align(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 2:
        output_result_tabular(argv_out, res_d, query, q_offsets, q_names);
        output_result_fa(argv_out + ".fasta", res_d, query, q_offsets, q_names);
        break;
    case 3:
        output_result_cast(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 4:
        output_result_a3m(argv_out, res_d, query, q_offsets, q_names);
        break;
    case 5:
        output_result_reduce(argv_out, res_d, query, q_offsets, q_names);
        break;
    default:
        break;
    }

    free(query);

    gettimeofday(&t_end, NULL);

    cout << "Output Time:\t" << timeuse(t_start, t_end) << endl;

    cout << "Finished." << endl;
}